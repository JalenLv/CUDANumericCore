#include "cncblas.h"
#include <iostream>
#include <cstdio>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <string>

const int M = 1 << 10;
const int N = 1 << 11;
const int kl = 100;
const int ku = 120;
//const int M = 4;
//const int N = 3;
//const int kl = 1;
//const int ku = 1;

const float PI = 3.14159265358979323846;

int main() {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipComplex *alpha = new hipComplex(cncblasRandC);
  hipComplex *beta = new hipComplex(cncblasRandC);
//  hipComplex one = make_hipComplex(1.0, 0.0);
//  hipComplex *alpha = new hipComplex(one);
//  hipComplex *beta = new hipComplex(one);


  // cncblas - 0 based
  hipComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipComplex[nColsA * nRowsA];
  h_x_cnc = new hipComplex[N];
  h_y_cnc = new hipComplex[M];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(hipComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipComplex[M * N];
  h_x_gemv = new hipComplex[N];
  h_y_gemv = new hipComplex[M];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, M * sizeof(hipComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandC;
//      h_A_cnc[(row - col + ku) * nColsA + col] = one;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRandC;
//    h_x_cnc[i] = one;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRandC;
//    h_y_cnc[i] = one;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, M * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasCgbmv(CNCBLAS_OP_N, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasCgemv(CNCBLAS_OP_N, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, M * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < M; i++) {
    if (!cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) {
      std::cout << "Results do not match at " << i << std::endl;
    }
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
  return 0;
}
