#include "hip/hip_runtime.h"
#include "level_one.cuh"

/* -------------------- AXPY -------------------- */

const int BLOCK_SIZE = 256;

__global__ void cncblasSaxpyKernel(size_t n, float alpha, const float *x, float *y) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    y[idx] = alpha * x[idx] + y[idx];
  }
}

void cncblasSaxpy(size_t n, const float *alpha, const float *x, float *y) {
  size_t num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasSaxpyKernel<<<num_blocks, BLOCK_SIZE>>>(n, *alpha, x, y);
}

__global__ void cncblasDaxpyKernel(size_t n, double alpha, const double *x, double *y) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    y[idx] = alpha * x[idx] + y[idx];
  }
}

void cncblasDaxpy(size_t n, const double *alpha, const double *x, double *y) {
  size_t num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasDaxpyKernel<<<num_blocks, BLOCK_SIZE>>>(n, *alpha, x, y);
}

__global__ void cncblasCaxpyKernel(size_t n, hipComplex alpha, const hipComplex *x, hipComplex *y) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    y[idx] = hipCaddf(hipCmulf(alpha, x[idx]), y[idx]);
  }
}

void cncblasCaxpy(size_t n, const hipComplex *alpha, const hipComplex *x, hipComplex *y) {
  size_t num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasCaxpyKernel<<<num_blocks, BLOCK_SIZE>>>(n, *alpha, x, y);
}

__global__ void cncblasZaxpyKernel(size_t n, hipDoubleComplex alpha, const hipDoubleComplex *x, hipDoubleComplex *y) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    y[idx] = hipCadd(hipCmul(alpha, x[idx]), y[idx]);
  }
}

void cncblasZaxpy(size_t n, const hipDoubleComplex *alpha, const hipDoubleComplex *x, hipDoubleComplex *y) {
  size_t num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasZaxpyKernel<<<num_blocks, BLOCK_SIZE>>>(n, *alpha, x, y);
}
