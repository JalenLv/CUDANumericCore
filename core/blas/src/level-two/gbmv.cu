#include "hip/hip_runtime.h"
#include "cncblas.h"
#include "src/helpers.cuh"

/* -------------------- KERNEL DECLARATION --------------------- */

__global__ void cncblasSgbmvKernelN(int m, int ku, int nColsA, int nRowsA,
                                    const float *alpha, const float *A, const float *x, float *y);
__global__ void cncblasSgbmvKernelT(int m, int ku, int nRowsA, int nColsA,
                                    const float *alpha, const float *A, const float *x, float *y);
__global__ void cncblasDgbmvKernelN(int m, int ku, int nColsA, int nRowsA,
                                    const double *alpha, const double *A, const double *x, double *y);
__global__ void cncblasDgbmvKernelT(int m, int ku, int nRowsA, int nColsA,
                                    const double *alpha, const double *A, const double *x, double *y);
__global__ void cncblasCgbmvKernelN_phase1(int m, int ku, int nColsA, int nRowsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1);
__global__ void cncblasCgbmvKernelN_phase2(int m, const hipComplex *phase1, hipComplex *y);
__global__ void cncblasCgbmvKernelT_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1);
__global__ void cncblasCgbmvKernelT_phase2(int nColsA, const hipComplex *phase1, hipComplex *y);
__global__ void cncblasCgbmvKernelC_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1);
__global__ void cncblasCgbmvKernelC_phase2(int nColsA, const hipComplex *phase1, hipComplex *y);
__global__ void cncblasZgbmvKernelN_phase1(int m, int ku, int nColsA, int nRowsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1);
__global__ void cncblasZgbmvKernelN_phase2(int m, const hipDoubleComplex *phase1, hipDoubleComplex *y);
__global__ void cncblasZgbmvKernelT_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1);
__global__ void cncblasZgbmvKernelT_phase2(int nColsA, const hipDoubleComplex *phase1, hipDoubleComplex *y);
__global__ void cncblasZgbmvKernelC_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1);
__global__ void cncblasZgbmvKernelC_phase2(int nColsA, const hipDoubleComplex *phase1, hipDoubleComplex *y);

/* -------------------- GBMV --------------------- */

const int BLOCK_SIZE = 256;
const int WARP_SIZE = 32;

void cncblasSgbmv(cncblasOperation_t trans,
                  int m, int n, int kl, int ku,
                  const float *alpha, const float *A, const float *x,
                  const float *beta, float *y) {
  // Check the parameters
  gbmvParamErrorCheck(m, n, kl, ku, alpha, A, x, beta, y);
  if (trans == CNCBLAS_OP_C) {
    std::cerr << "CNCBLAS_OP_C is not supported" << std::endl;
    exit(1);
  }
  // Preprocess the scalar parameters
  float *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // Quick return if possible
  if (m == 0 || n == 0 || (*h_alpha == 0 && *h_beta == 1)) {
    return;
  }

  // Set LENY, the length of the vectors x and y
  int leny;
  if (trans == CNCBLAS_OP_N) {
    leny = m;
  } else {
    leny = n;
  }

  // First, form y = beta * y
  if (*h_beta != 1) {
    cncblasSscal(leny, d_beta, y);
  }

  // Form y = alpha * A * x + y
  const int nColsA = cncblasMin(n, m + ku);
  const int nRowsA = ku + kl + 1;
  if (trans == CNCBLAS_OP_N) {
    const dim3 GRID_SIZE((m + BLOCK_SIZE - 1) / BLOCK_SIZE, 32);
    cncblasSgbmvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nColsA, nRowsA, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, 32);
    cncblasSgbmvKernelT<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

void cncblasDgbmv(cncblasOperation_t trans,
                  int m, int n, int kl, int ku,
                  const double *alpha, const double *A, const double *x,
                  const double *beta, double *y) {
  // Check the parameters
  gbmvParamErrorCheck(m, n, kl, ku, alpha, A, x, beta, y);
  if (trans == CNCBLAS_OP_C) {
    std::cerr << "CNCBLAS_OP_C is not supported" << std::endl;
    exit(1);
  }
  // Preprocess the scalar parameters
  double *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // Quick return if possible
  if (m == 0 || n == 0 || (*h_alpha == 0 && *h_beta == 1)) {
    return;
  }

  // Set LENY, the length of the vectors x and y
  int leny;
  if (trans == CNCBLAS_OP_N) {
    leny = m;
  } else {
    leny = n;
  }

  // First, form y = beta * y
  if (*h_beta != 1) {
    cncblasDscal(leny, d_beta, y);
  }

  // Form y = alpha * A * x + y
  const int nColsA = cncblasMin(n, m + ku);
  const int nRowsA = ku + kl + 1;
  if (trans == CNCBLAS_OP_N) {
    const dim3 GRID_SIZE((m + BLOCK_SIZE - 1) / BLOCK_SIZE, 32);
    cncblasDgbmvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nColsA, nRowsA, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, 32);
    cncblasDgbmvKernelT<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

void cncblasCgbmv(cncblasOperation_t trans,
                  int m, int n, int kl, int ku,
                  const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                  const hipComplex *beta, hipComplex *y) {
  // Check the parameters
  gbmvParamErrorCheck(m, n, kl, ku, alpha, A, x, beta, y);
  // Preprocess the scalar parameters
  hipComplex *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // Quick return if possible
  hipComplex zero = make_hipComplex(0.0f, 0.0f);
  hipComplex one = make_hipComplex(1.0f, 0.0f);
  if (m == 0 || n == 0 || (cncblasComplexIsEqual(h_alpha, &zero) && cncblasComplexIsEqual(h_beta, &one))) {
    return;
  }

  // Set LENY, the length of the vectors x and y
  int leny;
  if (trans == CNCBLAS_OP_N) {
    leny = m;
  } else {
    leny = n;
  }

  // First, form y = beta * y
  if (!cncblasComplexIsEqual(h_beta, &one)) {
    cncblasCscal(leny, d_beta, y);
  }

  // Form y = alpha * A * x + y
  const int nColsA = cncblasMin(n, m + ku);
  const int nRowsA = ku + kl + 1;
  if (trans == CNCBLAS_OP_N) {
    const dim3 GRID_SIZE((m + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, m * GRID_SIZE.y * sizeof(hipComplex)));

    cncblasCgbmvKernelN_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nColsA, nRowsA, d_alpha, A, x, phase1);
    cncblasCgbmvKernelN_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(m, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else if (trans == CNCBLAS_OP_T) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, nColsA * GRID_SIZE.y * sizeof(hipComplex)));

    cncblasCgbmvKernelT_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, phase1);
    cncblasCgbmvKernelT_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(nColsA, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else if (trans == CNCBLAS_OP_C) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, nColsA * GRID_SIZE.y * sizeof(hipComplex)));

    cncblasCgbmvKernelC_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, phase1);
    cncblasCgbmvKernelC_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(nColsA, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

void cncblasZgbmv(cncblasOperation_t trans,
                  int m, int n, int kl, int ku,
                  const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                  const hipDoubleComplex *beta, hipDoubleComplex *y) {
  // Check the parameters
  gbmvParamErrorCheck(m, n, kl, ku, alpha, A, x, beta, y);
  // Preprocess the scalar parameters
  hipDoubleComplex *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // Quick return if possible
  hipDoubleComplex zero = make_hipDoubleComplex(0.0, 0.0);
  hipDoubleComplex one = make_hipDoubleComplex(1.0, 0.0);
  if (m == 0 || n == 0 || (cncblasComplexIsEqual(h_alpha, &zero) && cncblasComplexIsEqual(h_beta, &one))) {
    return;
  }

  // Set LENY, the length of the vectors x and y
  int leny;
  if (trans == CNCBLAS_OP_N) {
    leny = m;
  } else {
    leny = n;
  }

  // First, form y = beta * y
  if (!cncblasComplexIsEqual(h_beta, &one)) {
    cncblasZscal(leny, d_beta, y);
  }

  // Form y = alpha * A * x + y
  const int nColsA = cncblasMin(n, m + ku);
  const int nRowsA = ku + kl + 1;
  if (trans == CNCBLAS_OP_N) {
    const dim3 GRID_SIZE((m + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipDoubleComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, m * GRID_SIZE.y * sizeof(hipDoubleComplex)));

    cncblasZgbmvKernelN_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nColsA, nRowsA, d_alpha, A, x, phase1);
    cncblasZgbmvKernelN_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(m, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else if (trans == CNCBLAS_OP_T) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipDoubleComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, nColsA * GRID_SIZE.y * sizeof(hipDoubleComplex)));

    cncblasZgbmvKernelT_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, phase1);
    cncblasZgbmvKernelT_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(nColsA, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else if (trans == CNCBLAS_OP_C) {
    const dim3 GRID_SIZE((nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE, WARP_SIZE);

    hipDoubleComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, nColsA * GRID_SIZE.y * sizeof(hipDoubleComplex)));

    cncblasZgbmvKernelC_phase1<<<GRID_SIZE, BLOCK_SIZE>>>
            (m, ku, nRowsA, nColsA, d_alpha, A, x, phase1);
    cncblasZgbmvKernelC_phase2<<<GRID_SIZE.x, BLOCK_SIZE>>>(nColsA, phase1, y);

    checkCudaErrors(hipFree(phase1));
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

/* -------------------- KERNEL DEFINITION --------------------- */

__global__ void cncblasSgbmvKernelN(int m, int ku, int nColsA, int nRowsA,
                                    const float *alpha, const float *A, const float *x, float *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int offset = ku - row;
  int col = idx + offset;

  if (idx < m) {
    float temp = 0.0f;
    while (row < nRowsA && col >= 0) {
      if (col < nColsA && col >= 0) {
        temp += A[row * nColsA + col] * x[col];
      }
      row += gridDim.y;
      col -= gridDim.y;
    }

    atomicAdd(&y[idx], *alpha * temp);
  }
}

__global__ void cncblasSgbmvKernelT(int m, int ku, int nRowsA, int nColsA,
                                    const float *alpha, const float *A, const float *x, float *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    float temp = 0.0f;
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp += A[row * nColsA + col] * x[i];
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    atomicAdd(&y[col], *alpha * temp);
  }
}

__global__ void cncblasDgbmvKernelN(int m, int ku, int nColsA, int nRowsA,
                                    const double *alpha, const double *A, const double *x, double *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int offset = ku - row;
  int col = idx + offset;

  if (idx < m) {
    double temp = 0.0f;
    while (row < nRowsA && col >= 0) {
      if (col < nColsA && col >= 0) {
        temp += A[row * nColsA + col] * x[col];
      }
      row += gridDim.y;
      col -= gridDim.y;
    }

    atomicAdd(&y[idx], *alpha * temp);
  }
}

__global__ void cncblasDgbmvKernelT(int m, int ku, int nRowsA, int nColsA,
                                    const double *alpha, const double *A, const double *x, double *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    double temp = 0.0f;
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp += A[row * nColsA + col] * x[i];
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    atomicAdd(&y[col], *alpha * temp);
  }
}

__global__ void cncblasCgbmvKernelN_phase1(int m, int ku, int nColsA, int nRowsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int offset = ku - row;
  int col = idx + offset;

  if (tid < m) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    while (row < nRowsA && col >= 0) {
      if (col < nColsA && col >= 0) {
        temp = hipCaddf(temp, hipCmulf(A[row * nColsA + col], x[col]));
      }
      row += gridDim.y;
      col -= gridDim.y;
    }

    phase1[blockIdx.y * m + idx] = hipCmulf(*alpha, temp);
  }
}

__global__ void cncblasCgbmvKernelN_phase2(int m, const hipComplex *phase1, hipComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < m) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCaddf(temp, phase1[i * m + idx]);
    }
    y[idx] = hipCaddf(y[idx], temp);
  }
}

__global__ void cncblasCgbmvKernelT_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp = hipCaddf(temp, hipCmulf(A[row * nColsA + col], x[i]));
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    phase1[blockIdx.y * nColsA + idx] = hipCmulf(*alpha, temp);
  }
}

__global__ void cncblasCgbmvKernelT_phase2(int nColsA, const hipComplex *phase1, hipComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < nColsA) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCaddf(temp, phase1[i * nColsA + idx]);
    }
    y[idx] = hipCaddf(y[idx], temp);
  }
}

__global__ void cncblasCgbmvKernelC_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                           hipComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp = hipCaddf(temp, hipCmulf(hipConjf(A[row * nColsA + col]), x[i]));
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    phase1[blockIdx.y * nColsA + idx] = hipCmulf(*alpha, temp);
  }
}

__global__ void cncblasCgbmvKernelC_phase2(int nColsA, const hipComplex *phase1, hipComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < nColsA) {
    hipComplex temp = make_hipComplex(0.0f, 0.0f);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCaddf(temp, phase1[i * nColsA + idx]);
    }
    y[idx] = hipCaddf(y[idx], temp);
  }
}

__global__ void cncblasZgbmvKernelN_phase1(int m, int ku, int nColsA, int nRowsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int offset = ku - row;
  int col = idx + offset;

  if (tid < m) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    while (row < nRowsA && col >= 0) {
      if (col < nColsA && col >= 0) {
        temp = hipCadd(temp, hipCmul(A[row * nColsA + col], x[col]));
      }
      row += gridDim.y;
      col -= gridDim.y;
    }

    phase1[blockIdx.y * m + idx] = hipCmul(*alpha, temp);
  }
}

__global__ void cncblasZgbmvKernelN_phase2(int m, const hipDoubleComplex *phase1, hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < m) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCadd(temp, phase1[i * m + idx]);
    }
    y[idx] = hipCadd(y[idx], temp);
  }
}

__global__ void cncblasZgbmvKernelT_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp = hipCadd(temp, hipCmul(A[row * nColsA + col], x[i]));
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    phase1[blockIdx.y * nColsA + idx] = hipCmul(*alpha, temp);
  }
}

__global__ void cncblasZgbmvKernelT_phase2(int nColsA, const hipDoubleComplex *phase1, hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < nColsA) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCadd(temp, phase1[i * nColsA + idx]);
    }
    y[idx] = hipCadd(y[idx], temp);
  }
}

__global__ void cncblasZgbmvKernelC_phase1(int m, int ku, int nRowsA, int nColsA,
                                           const hipDoubleComplex *alpha, const hipDoubleComplex *A,
                                           const hipDoubleComplex *x,
                                           hipDoubleComplex *phase1) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;
  int row = blockIdx.y;
  int col = idx;
  int offset = row - ku;

  if (idx < nColsA) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    while (row < nRowsA) {
      int i = col + offset;
      if (i >= 0 && i < m) {
        temp = hipCadd(temp, hipCmul(hipConj(A[row * nColsA + col]), x[i]));
      }
      row += gridDim.y;
      offset += gridDim.y;
    }

    phase1[blockIdx.y * nColsA + idx] = hipCmul(*alpha, temp);
  }
}

__global__ void cncblasZgbmvKernelC_phase2(int nColsA, const hipDoubleComplex *phase1, hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + tid;

  if (idx < nColsA) {
    hipDoubleComplex temp = make_hipDoubleComplex(0.0, 0.0);
    for (int i = 0; i < WARP_SIZE; i++) {
      temp = hipCadd(temp, phase1[i * nColsA + idx]);
    }
    y[idx] = hipCadd(y[idx], temp);
  }
}
