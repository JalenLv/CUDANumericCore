#include "hip/hip_runtime.h"
#include "cncblas.h"
#include "src/helpers.cuh"

/* -------------------- KERNEL DECLARATION -------------------- */

__global__ void cncblasSgemvKernelN(const int lenx, const int leny,
                                    const float *alpha, const float *A, const float *x, float *y);
__global__ void cncblasSgemvKernelT(const int lenx, const int leny,
                                    const float *alpha, const float *A, const float *x, float *y);
__global__ void cncblasDgemvKernelN(const int lenx, const int leny,
                                    const double *alpha, const double *A, const double *x, double *y);
__global__ void cncblasDgemvKernelT(const int lenx, const int leny,
                                    const double *alpha, const double *A, const double *x, double *y);
__global__ void cncblasCgemvKernelN(const int lenx, const int leny,
                                    const hipComplex *alpha, const hipComplex *A, const hipComplex *x, hipComplex *y);
__global__ void cncblasCgemvKernelT_1(const int lenx, const int leny,
                                      const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                      hipComplex *phase1);
__global__ void cncblasCgemvKernelT_2(const int n, const hipComplex *phase1, hipComplex *y);
__global__ void cncblasCgemvKernelC_1(const int lenx, const int leny,
                                      const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                      hipComplex *phase1);
__global__ void cncblasCgemvKernelC_2(const int n, const hipComplex *phase1, hipComplex *y);
__global__ void cncblasZgemvKernelN(const int lenx, const int leny,
                                    const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                    hipDoubleComplex *y);
__global__ void cncblasZgemvKernelT_1(const int lenx, const int leny,
                                      const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                      hipDoubleComplex *y);
__global__ void cncblasZgemvKernelT_2(const int n, const hipDoubleComplex *phase1, hipDoubleComplex *y);
__global__ void cncblasZgemvKernelC_1(const int lenx, const int leny,
                                      const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                      hipDoubleComplex *y);
__global__ void cncblasZgemvKernelC_2(const int n, const hipDoubleComplex *phase1, hipDoubleComplex *y);

/* -------------------- GEMV -------------------- */

const int BLOCK_SIZE = 256;
const int WARP_SIZE = 32;

void cncblasSgemv(cncblasOperation_t trans,
                  int m, int n,
                  const float *alpha, const float *A, const float *x,
                  const float *beta, float *y) {
  // Test for invalid parameters
  gemvParamErrorCheck(m, n, alpha, A, x, beta, y);
  // Preprocess scalar pointers
  float *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // quick return if possible
  if (m == 0 || n == 0 || (*h_alpha == 0 && *h_beta == 1)) {
    return;
  }

  // Set `lenx` and `leny` based on the value of `trans`.
  // `lenx` and `leny` are the lengths of the vectors `x` and `y` respectively.
  int lenx = 0, leny = 0;
  if (trans == CNCBLAS_OP_N) {
    lenx = n;
    leny = m;
  } else {
    lenx = m;
    leny = n;
  }

  // First form y = beta * y
  if (*h_beta != 1)
    cncblasSscal(leny, d_beta, y);
  if (*h_alpha == 0) return;

  // Form y = alpha * op(A) * x + y
  if (trans == CNCBLAS_OP_N) {
    // Form y = alpha * A * x + y
    int GRID_SIZE = m;
    cncblasSgemvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    // Form y = alpha * A^T * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    cncblasSgemvKernelT<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }

  // Free memory
  free(h_alpha);
  free(h_beta);
  checkCudaErrors(hipFree(d_alpha));
  checkCudaErrors(hipFree(d_beta));
}

void cncblasDgemv(cncblasOperation_t trans,
                  int m, int n,
                  const double *alpha, const double *A, const double *x,
                  const double *beta, double *y) {
  // Test for invalid parameters
  gemvParamErrorCheck(m, n, alpha, A, x, beta, y);
  // Preprocess scalar pointers
  double *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  // quick return if possible
  if (m == 0 || n == 0 || (*h_alpha == 0 && *h_beta == 1)) {
    return;
  }

  // Set `lenx` and `leny` based on the value of `trans`.
  // `lenx` and `leny` are the lengths of the vectors `x` and `y` respectively.
  int lenx = 0, leny = 0;
  if (trans == CNCBLAS_OP_N) {
    lenx = n;
    leny = m;
  } else {
    lenx = m;
    leny = n;
  }

  // First form y = beta * y
  if (*h_beta != 1)
    cncblasDscal(leny, d_beta, y);
  if (*h_alpha == 0) return;

  // Form y = alpha * op(A) * x + y
  if (trans == CNCBLAS_OP_N) {
    // Form y = alpha * A * x + y
    int GRID_SIZE = m;
    cncblasDgemvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    // Form y = alpha * A^T * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    cncblasDgemvKernelT<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }

  // Free memory
  free(h_alpha);
  free(h_beta);
  checkCudaErrors(hipFree(d_alpha));
  checkCudaErrors(hipFree(d_beta));
}

void cncblasCgemv(cncblasOperation_t trans,
                  int m, int n,
                  const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                  const hipComplex *beta, hipComplex *y) {
  // Test for invalid parameters
  gemvParamErrorCheck(m, n, alpha, A, x, beta, y);
  // Preprocess scalar pointers
  hipComplex *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  hipComplex one = make_hipComplex(1, 0);
  hipComplex zero = make_hipComplex(0, 0);
  // quick return if possible
  if (m == 0 || n == 0 || (cncblasComplexIsEqual(h_alpha, &zero) && cncblasComplexIsEqual(h_beta, &one))) {
    return;
  }

  // Set `lenx` and `leny` based on the value of `trans`.
  // `lenx` and `leny` are the lengths of the vectors `x` and `y` respectively.
  int lenx = 0, leny = 0;
  if (trans == CNCBLAS_OP_N) {
    lenx = n;
    leny = m;
  } else {
    lenx = m;
    leny = n;
  }

  // First form y = beta * y
  if (!cncblasComplexIsEqual(h_beta, &one))
    cncblasCscal(leny, d_beta, y);
  if (cncblasComplexIsEqual(h_alpha, &zero)) return;

  // Form y = alpha * op(A) * x + y
  if (trans == CNCBLAS_OP_N) {
    // Form y = alpha * A * x + y
    int GRID_SIZE = m;
    cncblasCgemvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    // Form y = alpha * A^T * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    hipComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, leny * GRID_SIZE.y * sizeof(hipComplex)));
    cncblasCgemvKernelT_1<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, phase1);
    cncblasCgemvKernelT_2<<<leny, WARP_SIZE>>>(GRID_SIZE.y, phase1, y);
  } else if (trans == CNCBLAS_OP_C) {
    // Form y = alpha * A^H * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    hipComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, leny * GRID_SIZE.y * sizeof(hipComplex)));
    cncblasCgemvKernelC_1<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, phase1);
    cncblasCgemvKernelC_2<<<leny, WARP_SIZE>>>(GRID_SIZE.y, phase1, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

void cncblasZgemv(cncblasOperation_t trans,
                  int m, int n,
                  const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                  const hipDoubleComplex *beta, hipDoubleComplex *y) {
  // Test for invalid parameters
  gemvParamErrorCheck(m, n, alpha, A, x, beta, y);
  // Preprocess scalar pointers
  hipDoubleComplex *h_alpha, *h_beta, *d_alpha, *d_beta;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);
  cncblasScalarPointerPreprocess(beta, h_beta, d_beta);

  hipDoubleComplex one = make_hipDoubleComplex(1, 0);
  hipDoubleComplex zero = make_hipDoubleComplex(0, 0);
  // quick return if possible
  if (m == 0 || n == 0 || (cncblasComplexIsEqual(h_alpha, &zero) && cncblasComplexIsEqual(h_beta, &one))) {
    return;
  }

  // Set `lenx` and `leny` based on the value of `trans`.
  // `lenx` and `leny` are the lengths of the vectors `x` and `y` respectively.
  int lenx = 0, leny = 0;
  if (trans == CNCBLAS_OP_N) {
    lenx = n;
    leny = m;
  } else {
    lenx = m;
    leny = n;
  }

  // First form y = beta * y
  if (!cncblasComplexIsEqual(h_beta, &one))
    cncblasZscal(leny, d_beta, y);
  if (cncblasComplexIsEqual(h_alpha, &zero)) return;

  // Form y = alpha * op(A) * x + y
  if (trans == CNCBLAS_OP_N) {
    // Form y = alpha * A * x + y
    int GRID_SIZE = m;
    cncblasZgemvKernelN<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, y);
  } else if (trans == CNCBLAS_OP_T) {
    // Form y = alpha * A^T * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    hipDoubleComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, leny * GRID_SIZE.y * sizeof(hipDoubleComplex)));
    cncblasZgemvKernelT_1<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, phase1);
    cncblasZgemvKernelT_2<<<leny, WARP_SIZE>>>(GRID_SIZE.y, phase1, y);
  } else if (trans == CNCBLAS_OP_C) {
    // Form y = alpha * A^H * x + y
    dim3 GRID_SIZE((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    hipDoubleComplex *phase1;
    checkCudaErrors(hipMalloc(&phase1, leny * GRID_SIZE.y * sizeof(hipDoubleComplex)));
    cncblasZgemvKernelC_1<<<GRID_SIZE, BLOCK_SIZE>>>(lenx, leny, d_alpha, A, x, phase1);
    cncblasZgemvKernelC_2<<<leny, WARP_SIZE>>>(GRID_SIZE.y, phase1, y);
  } else {
    std::cerr << "Invalid value for `trans`" << std::endl;
    exit(1);
  }
}

/* -------------------- KERNEL DEFINITION -------------------- */

__device__ void cncblasSgemvWarpRdN(volatile float *sdata, int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

__global__ void cncblasSgemvKernelN(const int lenx, const int leny,
                                    const float *alpha, const float *A, const float *x, float *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ float sdata[BLOCK_SIZE];
  sdata[tid] = 0;
  __syncthreads();
  int i = tid;
  while (i < lenx) {
    sdata[tid] += A[row * lenx + i] * x[i];
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  if (tid < 128) sdata[tid] += sdata[tid + 128];
  __syncthreads();
  if (tid < 64) sdata[tid] += sdata[tid + 64];
  __syncthreads();
  if (tid < 32) cncblasSgemvWarpRdN(sdata, tid);

  if (tid == 0) y[blockIdx.x] += *alpha * sdata[0];
}

__global__ void cncblasSgemvKernelT(const int lenx, const int leny,
                                    const float *alpha, const float *A, const float *x, float *y) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ float sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : 0;
  __syncthreads();

  float temp = 0;
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp += A[(grow + row) * leny + gcol] * sdata[row];
    }
  }

  if (gcol < leny)
    atomicAdd(&y[gcol], *alpha * temp);
}

__device__ void cncblasDgemvWarpRdN(volatile double *sdata, int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

__global__ void cncblasDgemvKernelN(const int lenx, const int leny,
                                    const double *alpha, const double *A, const double *x, double *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ double sdata[BLOCK_SIZE];
  sdata[tid] = 0;
  __syncthreads();
  int i = tid;
  while (i < lenx) {
    sdata[tid] += A[row * lenx + i] * x[i];
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  if (tid < 128) sdata[tid] += sdata[tid + 128];
  __syncthreads();
  if (tid < 64) sdata[tid] += sdata[tid + 64];
  __syncthreads();
  if (tid < 32) cncblasDgemvWarpRdN(sdata, tid);

  if (tid == 0) y[blockIdx.x] += *alpha * sdata[0];
}

__global__ void cncblasDgemvKernelT(const int lenx, const int leny,
                                    const double *alpha, const double *A, const double *x, double *y) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ double sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : 0;
  __syncthreads();

  double temp = 0;
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp += A[(grow + row) * leny + gcol] * sdata[row];
    }
  }

  if (gcol < leny)
    atomicAdd(&y[gcol], *alpha * temp);
}

__device__ void cncblasCgemvWarpRdN(volatile hipComplex *sdata, int tid) {
  volatile hipComplex *temp = sdata + tid;
  cncblasCVaddf(temp, temp + 32);
  cncblasCVaddf(temp, temp + 16);
  cncblasCVaddf(temp, temp + 8);
  cncblasCVaddf(temp, temp + 4);
  cncblasCVaddf(temp, temp + 2);
  cncblasCVaddf(temp, temp + 1);
}

__global__ void cncblasCgemvKernelN(const int lenx, const int leny,
                                    const hipComplex *alpha, const hipComplex *A, const hipComplex *x, hipComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipComplex sdata[BLOCK_SIZE];
  sdata[tid] = make_hipComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < lenx) {
    sdata[tid] = hipCaddf(sdata[tid], hipCmulf(A[row * lenx + i], x[i]));
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  if (tid < 128) sdata[tid] = hipCaddf(sdata[tid], sdata[tid + 128]);
  __syncthreads();
  if (tid < 64) sdata[tid] = hipCaddf(sdata[tid], sdata[tid + 64]);
  __syncthreads();
  if (tid < 32) cncblasCgemvWarpRdN(sdata, tid);

  if (tid == 0) y[blockIdx.x] = hipCaddf(y[blockIdx.x], hipCmulf(*alpha, sdata[0]));
}

__global__ void cncblasCgemvKernelT_1(const int lenx, const int leny,
                                      const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                      hipComplex *phase1) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ hipComplex sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : make_hipComplex(0, 0);
  __syncthreads();

  hipComplex temp = make_hipComplex(0, 0);
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp = hipCaddf(temp, hipCmulf(A[(grow + row) * leny + gcol], sdata[row]));
    }
  }

  if (gcol < leny)
    phase1[gcol * gridDim.y + blockIdx.y] = hipCmulf(*alpha, temp);
}

__device__ void cncblasCgemvWarpRdT(volatile hipComplex *sdata, int tid) {
  volatile hipComplex *temp = sdata + tid;
  cncblasCVaddf(temp, temp + 16);
  cncblasCVaddf(temp, temp + 8);
  cncblasCVaddf(temp, temp + 4);
  cncblasCVaddf(temp, temp + 2);
  cncblasCVaddf(temp, temp + 1);
}

__global__ void cncblasCgemvKernelT_2(const int n, const hipComplex *phase1, hipComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipComplex sdata[WARP_SIZE];
  sdata[tid] = make_hipComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < n) {
    sdata[tid] = hipCaddf(sdata[tid], phase1[row * n + i]);
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  cncblasCgemvWarpRdT(sdata, tid);

  if (tid == 0) y[row] = hipCaddf(y[row], sdata[0]);
}

__global__ void cncblasCgemvKernelC_1(const int lenx, const int leny,
                                      const hipComplex *alpha, const hipComplex *A, const hipComplex *x,
                                      hipComplex *phase1) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ hipComplex sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : make_hipComplex(0, 0);
  __syncthreads();

  hipComplex temp = make_hipComplex(0, 0);
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp = hipCaddf(temp, hipCmulf(hipConjf(A[(grow + row) * leny + gcol]), sdata[row]));
    }
  }

  if (gcol < leny)
    phase1[gcol * gridDim.y + blockIdx.y] = hipCmulf(*alpha, temp);
}

__global__ void cncblasCgemvKernelC_2(const int n, const hipComplex *phase1, hipComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipComplex sdata[WARP_SIZE];
  sdata[tid] = make_hipComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < n) {
    sdata[tid] = hipCaddf(sdata[tid], phase1[row * n + i]);
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  cncblasCgemvWarpRdT(sdata, tid);

  if (tid == 0) y[row] = hipCaddf(y[row], sdata[0]);
}

__device__ void cncblasZgemvWarpRdN(volatile hipDoubleComplex *sdata, int tid) {
  volatile hipDoubleComplex *temp = sdata + tid;
  cncblasZVadd(temp, temp + 32);
  cncblasZVadd(temp, temp + 16);
  cncblasZVadd(temp, temp + 8);
  cncblasZVadd(temp, temp + 4);
  cncblasZVadd(temp, temp + 2);
  cncblasZVadd(temp, temp + 1);
}

__global__ void cncblasZgemvKernelN(const int lenx, const int leny,
                                    const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                    hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipDoubleComplex sdata[BLOCK_SIZE];
  sdata[tid] = make_hipDoubleComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < lenx) {
    sdata[tid] = hipCadd(sdata[tid], hipCmul(A[row * lenx + i], x[i]));
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  if (tid < 128) sdata[tid] = hipCadd(sdata[tid], sdata[tid + 128]);
  __syncthreads();
  if (tid < 64) sdata[tid] = hipCadd(sdata[tid], sdata[tid + 64]);
  __syncthreads();
  if (tid < 32) cncblasZgemvWarpRdN(sdata, tid);

  if (tid == 0) y[blockIdx.x] = hipCadd(y[blockIdx.x], hipCmul(*alpha, sdata[0]));
}

__global__ void cncblasZgemvKernelT_1(const int lenx, const int leny,
                                      const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                      hipDoubleComplex *phase1) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ hipDoubleComplex sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : make_hipDoubleComplex(0, 0);
  __syncthreads();

  hipDoubleComplex temp = make_hipDoubleComplex(0, 0);
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp = hipCadd(temp, hipCmul(A[(grow + row) * leny + gcol], sdata[row]));
    }
  }

  if (gcol < leny)
    phase1[gcol * gridDim.y + blockIdx.y] = hipCmul(*alpha, temp);
}

__device__ void cncblasZgemvWarpRdT(volatile hipDoubleComplex *sdata, int tid) {
  volatile hipDoubleComplex *temp = sdata + tid;
  cncblasZVadd(temp, temp + 16);
  cncblasZVadd(temp, temp + 8);
  cncblasZVadd(temp, temp + 4);
  cncblasZVadd(temp, temp + 2);
  cncblasZVadd(temp, temp + 1);
}

__global__ void cncblasZgemvKernelT_2(const int n, const hipDoubleComplex *phase1, hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipDoubleComplex sdata[WARP_SIZE];
  sdata[tid] = make_hipDoubleComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < n) {
    sdata[tid] = hipCadd(sdata[tid], phase1[row * n + i]);
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  cncblasZgemvWarpRdT(sdata, tid);

  if (tid == 0) y[row] = hipCadd(y[row], sdata[0]);
}

__global__ void cncblasZgemvKernelC_1(const int lenx, const int leny,
                                      const hipDoubleComplex *alpha, const hipDoubleComplex *A, const hipDoubleComplex *x,
                                      hipDoubleComplex *phase1) {
  int grow = blockIdx.y * BLOCK_SIZE;
  int gcol = blockIdx.x * BLOCK_SIZE + threadIdx.x;

  __shared__ hipDoubleComplex sdata[BLOCK_SIZE];
  sdata[threadIdx.x] = (grow + threadIdx.x < lenx) ? x[grow + threadIdx.x] : make_hipDoubleComplex(0, 0);
  __syncthreads();

  hipDoubleComplex temp = make_hipDoubleComplex(0, 0);
  for (int row = 0; row < BLOCK_SIZE; row++) {
    if (grow + row < lenx && gcol < leny) {
      temp = hipCadd(temp, hipCmul(hipConj(A[(grow + row) * leny + gcol]), sdata[row]));
    }
  }

  if (gcol < leny)
    phase1[gcol * gridDim.y + blockIdx.y] = hipCmul(*alpha, temp);
}

__global__ void cncblasZgemvKernelC_2(const int n, const hipDoubleComplex *phase1, hipDoubleComplex *y) {
  int tid = threadIdx.x;
  int row = blockIdx.x;
  int stride = blockDim.x;

  __shared__ hipDoubleComplex sdata[WARP_SIZE];
  sdata[tid] = make_hipDoubleComplex(0, 0);
  __syncthreads();
  int i = tid;
  while (i < n) {
    sdata[tid] = hipCadd(sdata[tid], phase1[row * n + i]);
    i += stride;
  }
  __syncthreads();

  // Reduce the sum with loop unrolling
  cncblasZgemvWarpRdT(sdata, tid);

  if (tid == 0) y[row] = hipCadd(y[row], sdata[0]);
}

