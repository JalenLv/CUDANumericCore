#include "hip/hip_runtime.h"
#include "../cncblas.h"
#include <iostream>
#include <stdexcept>

/* -------------------- SCAL -------------------- */

const size_t BLOCK_SIZE = 256;

__global__ void cncblasSscalKernel(size_t n, const float *alpha, float *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] *= *alpha;
  }
}

void cncblasSscal(size_t n, const float *alpha, float *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasSscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasSscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasSscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasSscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}

__global__ void cncblasDscalKernel(size_t n, const double *alpha, double *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] *= *alpha;
  }
}

void cncblasDscal(size_t n, const double *alpha, double *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasDscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasDscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasDscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasDscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}

__global__ void cncblasCscalKernel(size_t n, const hipComplex *alpha, hipComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmulf(x[i], *alpha);
  }
}

void cncblasCscal(size_t n, const hipComplex *alpha, hipComplex *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasCscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasCscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasCscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasCscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}

__global__ void cncblasCsscalKernel(size_t n, const float *alpha, hipComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmulf(x[i], make_hipComplex(*alpha, 0.0f));
  }
}

void cncblasCsscal(size_t n, const float *alpha, hipComplex *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasCsscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasCsscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasCsscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasCsscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}

__global__ void cncblasZscalKernel(size_t n, const hipDoubleComplex *alpha, hipDoubleComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmul(x[i], *alpha);
  }
}

void cncblasZscal(size_t n, const hipDoubleComplex *alpha, hipDoubleComplex *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasZscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasZscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasZscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasZscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}

__global__ void cncblasZdscalKernel(size_t n, const double *alpha, hipDoubleComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmul(x[i], make_hipDoubleComplex(*alpha, 0.0));
  }
}

void cncblasZdscal(size_t n, const double *alpha, hipDoubleComplex *x) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("cncblasZdscal: invalid n");
    }
    if (alpha == nullptr) {
      throw std::invalid_argument("cncblasZdscal: alpha is null");
    }
    if (x == nullptr) {
      throw std::invalid_argument("cncblasZdscal: x is null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasZdscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, alpha, x);
}
