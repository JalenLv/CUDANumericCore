#include "hip/hip_runtime.h"
#include "../cncblas.h"
#include "helpers.cuh"

/* -------------------- KERNEL DECLARATION -------------------- */

__global__ void cncblasSgerKernel(int m, int n,
                                  const float *alpha, const float *x, const float *y,
                                  float *A);
__global__ void cncblasDgerKernel(int m, int n,
                                  const double *alpha, const double *x, const double *y,
                                  double *A);
__global__ void cncblasCgeruKernel(int m, int n,
                                   const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                                   hipComplex *A);
__global__ void cncblasCgercKernel(int m, int n,
                                   const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                                   hipComplex *A);
__global__ void cncblasZgeruKernel(int m, int n,
                                   const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                                   hipDoubleComplex *A);
__global__ void cncblasZgercKernel(int m, int n,
                                   const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                                   hipDoubleComplex *A);

/* -------------------- GER -------------------- */

void cncblasSger(int m, int n,
                 const float *alpha, const float *x, const float *y,
                 float *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  float *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  if (m == 0 || n == 0 || *h_alpha == 0) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasSgerKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

void cncblasDger(int m, int n,
                 const double *alpha, const double *x, const double *y,
                 double *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  double *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  if (m == 0 || n == 0 || *h_alpha == 0) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasDgerKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

void cncblasCgeru(int m, int n,
                  const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                  hipComplex *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  hipComplex *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  hipComplex zero = make_hipComplex(0, 0);
  if (m == 0 || n == 0 || cncblasComplexIsEqual(h_alpha, &zero)) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasCgeruKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

void cncblasCgerc(int m, int n,
                  const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                  hipComplex *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  hipComplex *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  hipComplex zero = make_hipComplex(0, 0);
  if (m == 0 || n == 0 || cncblasComplexIsEqual(h_alpha, &zero)) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasCgeruKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

void cncblasZgeru(int m, int n,
                  const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                  hipDoubleComplex *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  hipDoubleComplex *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  hipDoubleComplex zero = make_hipDoubleComplex(0, 0);
  if (m == 0 || n == 0 || cncblasComplexIsEqual(h_alpha, &zero)) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasZgeruKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

void cncblasZgerc(int m, int n,
                  const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                  hipDoubleComplex *A) {
  // check for invalid arguments
  gerParamErrorCheck(m, n, alpha, x, y, A);
  // allocate memory for scalar pointers
  hipDoubleComplex *h_alpha, *d_alpha;
  gerScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // quick return if possible
  hipDoubleComplex zero = make_hipDoubleComplex(0, 0);
  if (m == 0 || n == 0 || cncblasComplexIsEqual(h_alpha, &zero)) {
    return;
  }

  // launch kernel
  dim3 BLOCK_SIZE(32, 16);
  dim3 GRID_SIZE((n + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
                 (m + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);
  cncblasZgercKernel<<<GRID_SIZE, BLOCK_SIZE>>>(m, n, d_alpha, x, y, A);
}

/* -------------------- KERNEL DEFINITION -------------------- */

__global__ void cncblasSgerKernel(int m, int n,
                                  const float *alpha, const float *x, const float *y,
                                  float *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    float a = A[row * n + col];
    float x_val = x[row];
    float y_val = y[col];
    A[row * n + col] = *alpha * x_val * y_val + a;
  }
}

__global__ void cncblasDgerKernel(int m, int n,
                                  const double *alpha, const double *x, const double *y,
                                  double *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    double a = A[row * n + col];
    double x_val = x[row];
    double y_val = y[col];
    A[row * n + col] = *alpha * x_val * y_val + a;
  }
}

__global__ void cncblasCgeruKernel(int m, int n,
                                   const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                                   hipComplex *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    hipComplex a = A[row * n + col];
    hipComplex x_val = x[row];
    hipComplex y_val = y[col];
    A[row * n + col] = hipCaddf(hipCmulf(*alpha, hipCmulf(x_val, y_val)), a);
  }
}

__global__ void cncblasCgercKernel(int m, int n,
                                   const hipComplex *alpha, const hipComplex *x, const hipComplex *y,
                                   hipComplex *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    hipComplex a = A[row * n + col];
    hipComplex x_val = x[row];
    hipComplex y_val = y[col];
    A[row * n + col] = hipCaddf(hipCmulf(*alpha, hipCmulf(x_val, hipConjf(y_val))), a);
  }
}

__global__ void cncblasZgeruKernel(int m, int n,
                                   const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                                   hipDoubleComplex *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    hipDoubleComplex a = A[row * n + col];
    hipDoubleComplex x_val = x[row];
    hipDoubleComplex y_val = y[col];
    A[row * n + col] = hipCadd(hipCmul(*alpha, hipCmul(x_val, y_val)), a);
  }
}

__global__ void cncblasZgercKernel(int m, int n,
                                   const hipDoubleComplex *alpha, const hipDoubleComplex *x, const hipDoubleComplex *y,
                                   hipDoubleComplex *A) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    hipDoubleComplex a = A[row * n + col];
    hipDoubleComplex x_val = x[row];
    hipDoubleComplex y_val = y[col];
    A[row * n + col] = hipCadd(hipCmul(*alpha, hipCmul(x_val, hipConj(y_val))), a);
  }
}

