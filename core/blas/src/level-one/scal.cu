#include "hip/hip_runtime.h"
#include "cncblas.h"
#include <iostream>
#include <stdexcept>
#include "src/helpers.cuh"

/* -------------------- SCAL -------------------- */

const size_t BLOCK_SIZE = 256;

__global__ void cncblasSscalKernel(size_t n, const float *alpha, float *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] *= *alpha;
  }
}

void cncblasSscal(size_t n, const float *alpha, float *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  float *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasSscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}

__global__ void cncblasDscalKernel(size_t n, const double *alpha, double *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] *= *alpha;
  }
}

void cncblasDscal(size_t n, const double *alpha, double *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  double *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasDscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}

__global__ void cncblasCscalKernel(size_t n, const hipComplex *alpha, hipComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmulf(x[i], *alpha);
  }
}

void cncblasCscal(size_t n, const hipComplex *alpha, hipComplex *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  hipComplex *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasCscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}

__global__ void cncblasCsscalKernel(size_t n, const float *alpha, hipComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmulf(x[i], make_hipComplex(*alpha, 0.0f));
  }
}

void cncblasCsscal(size_t n, const float *alpha, hipComplex *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  float *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasCsscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}

__global__ void cncblasZscalKernel(size_t n, const hipDoubleComplex *alpha, hipDoubleComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmul(x[i], *alpha);
  }
}

void cncblasZscal(size_t n, const hipDoubleComplex *alpha, hipDoubleComplex *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  hipDoubleComplex *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasZscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}

__global__ void cncblasZdscalKernel(size_t n, const double *alpha, hipDoubleComplex *x) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = hipCmul(x[i], make_hipDoubleComplex(*alpha, 0.0));
  }
}

void cncblasZdscal(size_t n, const double *alpha, hipDoubleComplex *x) {
  // Check for invalid inputs
  scalParamErrorCheck(n, alpha, x);
  // Preprocess the scalar parameters
  double *h_alpha, *d_alpha;
  cncblasScalarPointerPreprocess(alpha, h_alpha, d_alpha);

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasZdscalKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, d_alpha, x);
}
