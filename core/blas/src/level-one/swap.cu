#include "hip/hip_runtime.h"
#include "cncblas.h"
#include <iostream>
#include <stdexcept>

/* -------------------- SWAP -------------------- */

const size_t BLOCK_SIZE = 256;

template<typename T>
__global__ void cncblasSwapKernel(size_t n, T *x, T *y) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    T temp = x[i];
    x[i] = y[i];
    y[i] = temp;
  }
}

template<typename T>
void cncblasSwap(size_t n, T *x, T *y) {
  // Check for invalid inputs
  try {
    if (n <= 0) {
      throw std::invalid_argument("n must be greater than 0");
    }
    if (x == nullptr || y == nullptr) {
      throw std::invalid_argument("x and y must not be null");
    }
  } catch (const std::invalid_argument &e) {
    std::cerr << e.what() << std::endl;
    exit(1);
  }

  // Launch the kernel
  size_t GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  cncblasSwapKernel<<<GRID_SIZE, BLOCK_SIZE>>>(n, x, y);
}

// Explicit instantiations
template void cncblasSwap(size_t n, float *x, float *y);
template void cncblasSwap(size_t n, double *x, double *y);
template void cncblasSwap(size_t n, hipComplex *x, hipComplex *y);
template void cncblasSwap(size_t n, hipDoubleComplex *x, hipDoubleComplex *y);

void cncblasSswap(size_t n, float *x, float *y) {
  cncblasSwap<float>(n, x, y);
}

void cncblasDswap(size_t n, double *x, double *y) {
  cncblasSwap<double>(n, x, y);
}

void cncblasCswap(size_t n, hipComplex *x, hipComplex *y) {
  cncblasSwap<hipComplex>(n, x, y);
}

void cncblasZswap(size_t n, hipDoubleComplex *x, hipDoubleComplex *y) {
  cncblasSwap<hipDoubleComplex>(n, x, y);
}

