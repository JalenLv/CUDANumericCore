#include "cncblas.cuh"
#include <iostream>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define N (1 << 22)

int main() {
  hipDoubleComplex *x = new hipDoubleComplex[N];
  hipDoubleComplex *d_x;
  for (size_t i = 0; i < N; i++) {
    x[i] = make_hipDoubleComplex(-i, i);
  }
  hipMalloc(&d_x, N * sizeof(hipDoubleComplex));
  hipMemcpy(d_x, x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

  hipDoubleComplex *y = new hipDoubleComplex[N];
  hipDoubleComplex *d_y;
  for (size_t i = 0; i < N; i++) {
    y[i] = make_hipDoubleComplex(2 * i, 2 * i);
  }
  hipMalloc(&d_y, N * sizeof(hipDoubleComplex));
  hipMemcpy(d_y, y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

  hipDoubleComplex alpha = make_hipDoubleComplex(2.0, -2.0);
  cncblasZaxpy(N, &alpha, d_x, d_y);
  hipMemcpy(y, d_y, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < 10; i++) {
    std::cout << y[i].x << ", " << y[i].y << std::endl;
  }

  for (size_t i = 0; i < N; i++) {
    y[i] = make_hipDoubleComplex(2 * i, 2 * i);
  }
  hipMemset(d_y, 0, N * sizeof(hipDoubleComplex));
  hipMemcpy(d_y, y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZaxpy(handle, N, &alpha, d_x, 1, d_y, 1);
  hipMemcpy(y, d_y, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  for (size_t i = 0; i < 10; i++) {
    std::cout << y[i].x << ", " << y[i].y << std::endl;
  }

  return 0;
}