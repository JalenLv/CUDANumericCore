#include "cncblas.cuh"
#include <iostream>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>

#define N (1 << 22)

const float PI = 3.14159265358979323846;

int main() {
  // Test the SWAP function, and use cublas
  // to verify the results

  // Initialize the vectors
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (size_t i = 0; i < N; i++) {
    x[i] = i;
    y[i] = i + 1;
  }

  // Swap the vectors
  cncblasSswap(N, x, y);

  // Verify the results
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSswap(handle, N, x, 1, y, 1);
  hipblasDestroy(handle);

  bool passed = true;
  for (size_t i = 0; i < N; i++) {
    if (x[i] != i || y[i] != i + 1) {
      passed = false;
      break;
    }
  }

  if (passed) {
    std::cout << "PASSED" << std::endl;
  } else {
    std::cout << "FAILED" << std::endl;
  }

  return 0;
}