#include "cncblas.h"
#include <iostream>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>

#define N (1 << 22)

const float PI = 3.14159265358979323846;

int main() {
  // Test the SWAP function, and use cublas
  // to verify the results

  // Initialize the vectors
  float *x, *y;
  checkCudaErrors(hipMallocManaged(&x, N * sizeof(float)));
  checkCudaErrors(hipMallocManaged(&y, N * sizeof(float)));

  for (size_t i = 0; i < N; i++) {
    x[i] = i;
    y[i] = i + 1;
  }

  // Swap the vectors
  cncblasSswap(N, x, y);

  // Verify the results
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSswap(handle, N, x, 1, y, 1);
  hipblasDestroy(handle);

  bool passed = true;
  for (size_t i = 0; i < N; i++) {
    if (x[i] != i || y[i] != i + 1) {
      passed = false;
      break;
    }
  }

  if (passed) {
    std::cout << "PASSED" << std::endl;
  } else {
    std::cout << "FAILED" << std::endl;
  }

  hipPointerAttribute_t attributes;
  hipPointerGetAttributes(&attributes, x);
  if (attributes.type == hipMemoryTypeManaged) {
    std::cout << "x is managed memory" << std::endl;
  } else if (attributes.type == hipMemoryTypeDevice) {
    std::cout << "x is device memory" << std::endl;
  } else if (attributes.type == hipMemoryTypeHost) {
    std::cout << "x is host memory" << std::endl;
  }

  return 0;
}