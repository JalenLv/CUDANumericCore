#include "hip/hip_runtime.h"
#include "cncblas.cuh"
#include <iostream>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>

#define N (1 << 22)

int main() {
  hipComplex *cx, *cy;
  hipComplex *d_cx, *d_cy;

  cx = new hipComplex[N];
  cy = new hipComplex[N];

  for (int i = 0; i < N; i++) {
    cx[i].x = 1.0f;
    cx[i].y = 1.0f;
    cy[i].x = 2.0f;
    cy[i].y = 2.0f;
  }

  hipMalloc(&d_cx, N * sizeof(hipComplex));
  hipMalloc(&d_cy, N * sizeof(hipComplex));
  hipMemcpy(d_cx, cx, N * sizeof(hipComplex), hipMemcpyHostToDevice);
  hipMemcpy(d_cy, cy, N * sizeof(hipComplex), hipMemcpyHostToDevice);

  float result = 0.0f;
  result = cncblasCnrm2(N, d_cx);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float result_cublas = 0.0f;
  hipblasScnrm2(handle, N, d_cx, 1, &result_cublas);
  float epsilon = 1e-6;
  if (abs(result - result_cublas) < epsilon) {
    std::cout << "Test passed" << std::endl;
  } else {
    std::cout << "Test failed" << std::endl;
    std::cout << "cncblasCnrm2: " << std::fixed << std::setprecision(10) << result << std::endl;
    std::cout << "hipblasScnrm2: " << std::fixed << std::setprecision(10) << result_cublas << std::endl;
    std::cout << "Difference: " << abs(result - result_cublas) << std::endl;
  }

  hipComplex cresult;
  cresult = cncblasCdotc(N, d_cx, d_cx);

  hipComplex cresult_cublas;
  hipblasCdotc(handle, N, d_cx, 1, d_cx, 1, &cresult_cublas);
  if (abs(cresult.x - cresult_cublas.x) < epsilon && abs(cresult.y - cresult_cublas.y) < epsilon) {
    std::cout << "Test passed" << std::endl;
    std::cout << "cncblasCdotc: " << std::fixed << std::setprecision(10) << cresult.x << " + " << cresult.y << "i"
              << std::endl;
    std::cout << "hipblasCdotc: " << std::fixed << std::setprecision(10) << cresult_cublas.x << " + " << cresult_cublas.y
              << "i" << std::endl;
  } else {
    std::cout << "Test failed" << std::endl;
    std::cout << "cncblasCdotu: " << cresult.x << " + " << cresult.y << "i" << std::endl;
    std::cout << "hipblasCdotc: " << cresult_cublas.x << " + " << cresult_cublas.y << "i" << std::endl;
    std::cout << "Difference: " << abs(cresult.x - cresult_cublas.x) << " + " << abs(cresult.y - cresult_cublas.y)
              << "i" << std::endl;
  }

  return 0;
}