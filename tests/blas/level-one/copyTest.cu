#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(copy, singlePrecision) {
  float *h_x, *h_y_cnc, *h_y_cublas;
  float *d_x, *d_y_cnc, *d_y_cublas;

  h_x = new float[N];
  h_y_cnc = new float[N];
  h_y_cublas = new float[N];

  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute copy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasScopy(handle, N, d_x, 1, d_y_cublas, 1);

  // Compute copy on GPU using cncblas
  cncblasScopy(N, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(float), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_FLOAT_EQ(h_y_cublas[i], h_y_cnc[i]);
  }

  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(copy, doublePrecision) {
  double *h_x, *h_y_cnc, *h_y_cublas;
  double *d_x, *d_y_cnc, *d_y_cublas;

  h_x = new double[N];
  h_y_cnc = new double[N];
  h_y_cublas = new double[N];

  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute copy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDcopy(handle, N, d_x, 1, d_y_cublas, 1);

  // Compute copy on GPU using cncblas
  cncblasDcopy(N, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(double), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_DOUBLE_EQ(h_y_cublas[i], h_y_cnc[i]);
  }

  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(copy, complexSinglePrecision) {
  hipComplex *h_x, *h_y_cnc, *h_y_cublas;
  hipComplex *d_x, *d_y_cnc, *d_y_cublas;

  h_x = new hipComplex[N];
  h_y_cnc = new hipComplex[N];
  h_y_cublas = new hipComplex[N];

  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute copy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCcopy(handle, N, d_x, 1, d_y_cublas, 1);

  // Compute copy on GPU using cncblas
  cncblasCcopy(N, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cublas + i, h_y_cnc + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << "); but got: (" << h_y_cnc[i].x
                  << "," << h_y_cnc[i].y << ")" << std::endl << "Error: (" << std::abs(h_y_cublas[i].x - h_y_cnc[i].x)
                  << "," << std::abs(h_y_cublas[i].y - h_y_cnc[i].y) << ")" << std::endl;
  }

  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(copy, complexDoublePrecision) {
  hipDoubleComplex *h_x, *h_y_cnc, *h_y_cublas;
  hipDoubleComplex *d_x, *d_y_cnc, *d_y_cublas;

  h_x = new hipDoubleComplex[N];
  h_y_cnc = new hipDoubleComplex[N];
  h_y_cublas = new hipDoubleComplex[N];

  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute copy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZcopy(handle, N, d_x, 1, d_y_cublas, 1);

  // Compute copy on GPU using cncblas
  cncblasZcopy(N, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cublas + i, h_y_cnc + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << "); but got: (" << h_y_cnc[i].x
                  << "," << h_y_cnc[i].y << ")" << std::endl << "Error: (" << std::abs(h_y_cublas[i].x - h_y_cnc[i].x)
                  << "," << std::abs(h_y_cublas[i].y - h_y_cnc[i].y) << ")" << std::endl;
  }

  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}
