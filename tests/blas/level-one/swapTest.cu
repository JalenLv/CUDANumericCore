#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(swap, singlePrecision) {
  float *h_x, *h_y, *h_x_cublas, *h_y_cublas;
  float *d_x, *d_y, *d_x_cublas, *d_y_cublas;

  h_x = new float[N];
  h_y = new float[N];
  h_x_cublas = new float[N];
  h_y_cublas = new float[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
    h_y[i] = cncblasRandf;
    h_x_cublas[i] = h_x[i];
    h_y_cublas[i] = h_y[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(float), hipMemcpyHostToDevice));

  // Perform swap using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSswap(handle, N, d_x_cublas, 1, d_y_cublas, 1);

  // Perform swap using cncblas
  cncblasSswap(N, d_x, d_y);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x, d_x, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(float), hipMemcpyDeviceToHost));

// Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_FLOAT_EQ(h_x[i], h_x_cublas[i]);
    EXPECT_FLOAT_EQ(h_y[i], h_y_cublas[i]);
  }

  delete[] h_x;
  delete[] h_y;
  delete[] h_x_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_x_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(swap, doublePrecision) {
  double *h_x, *h_y, *h_x_cublas, *h_y_cublas;
  double *d_x, *d_y, *d_x_cublas, *d_y_cublas;

  h_x = new double[N];
  h_y = new double[N];
  h_x_cublas = new double[N];
  h_y_cublas = new double[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
    h_y[i] = cncblasRand;
    h_x_cublas[i] = h_x[i];
    h_y_cublas[i] = h_y[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(double), hipMemcpyHostToDevice));

  // Perform swap using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDswap(handle, N, d_x_cublas, 1, d_y_cublas, 1);

  // Perform swap using cncblas
  cncblasDswap(N, d_x, d_y);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x, d_x, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y, d_y, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(double), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_DOUBLE_EQ(h_x[i], h_x_cublas[i]);
    EXPECT_DOUBLE_EQ(h_y[i], h_y_cublas[i]);
  }

  delete[] h_x;
  delete[] h_y;
  delete[] h_x_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_x_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(swap, complexSinglePrecision) {
  hipComplex *h_x, *h_y, *h_x_cublas, *h_y_cublas;
  hipComplex *d_x, *d_y, *d_x_cublas, *d_y_cublas;

  h_x = new hipComplex[N];
  h_y = new hipComplex[N];
  h_x_cublas = new hipComplex[N];
  h_y_cublas = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_x_cublas[i] = h_x[i];
    h_y_cublas[i] = h_y[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform swap using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCswap(handle, N, d_x_cublas, 1, d_y_cublas, 1);

  // Perform swap using cncblas
  cncblasCswap(N, d_x, d_y);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x, d_x, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y, d_y, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x + i, h_x_cublas + i));
    EXPECT_TRUE(cncblasComplexIsEqual(h_y + i, h_y_cublas + i));
  }

  delete[] h_x;
  delete[] h_y;
  delete[] h_x_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_x_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(swap, complexDoublePrecision) {
  hipDoubleComplex *h_x, *h_y, *h_x_cublas, *h_y_cublas;
  hipDoubleComplex *d_x, *d_y, *d_x_cublas, *d_y_cublas;

  h_x = new hipDoubleComplex[N];
  h_y = new hipDoubleComplex[N];
  h_x_cublas = new hipDoubleComplex[N];
  h_y_cublas = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_x_cublas[i] = h_x[i];
    h_y_cublas[i] = h_y[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform swap using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZswap(handle, N, d_x_cublas, 1, d_y_cublas, 1);

  // Perform swap using cncblas
  cncblasZswap(N, d_x, d_y);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x, d_x, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y, d_y, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x + i, h_x_cublas + i));
    EXPECT_TRUE(cncblasComplexIsEqual(h_y + i, h_y_cublas + i));
  }

  delete[] h_x;
  delete[] h_y;
  delete[] h_x_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_x_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}
