#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(nrm2, singlePrecision) {
  float *h_x, *d_x;
  float *result_cnc, *result_cublas;

  h_x = new float[N];
  result_cublas = new float(0.0f);
  result_cnc = new float(1.0f);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute nrm2 on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSnrm2(handle, N, d_x, 1, result_cublas);

  // Compute nrm2 on GPU using cncblas
  *result_cnc = cncblasSnrm2(N, d_x);

  // Compare the results
  EXPECT_FLOAT_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(nrm2, doublePrecision) {
  double *h_x, *d_x;
  double *result_cnc, *result_cublas;

  h_x = new double[N];
  result_cublas = new double(0.0);
  result_cnc = new double(1.0);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute nrm2 on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDnrm2(handle, N, d_x, 1, result_cublas);

  // Compute nrm2 on GPU using cncblas
  *result_cnc = cncblasDnrm2(N, d_x);

  // Compare the results
  EXPECT_DOUBLE_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(nrm2, complexSinglePrecision) {
  hipComplex *h_x, *d_x;
  float *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  result_cublas = new float(0.0f);
  result_cnc = new float(1.0f);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute nrm2 on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasScnrm2(handle, N, d_x, 1, result_cublas);

  // Compute nrm2 on GPU using cncblas
  *result_cnc = cncblasCnrm2(N, d_x);

  // Compare the results
  EXPECT_FLOAT_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(nrm2, complexDoublePrecision) {
  hipDoubleComplex *h_x, *d_x;
  double *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  result_cublas = new double(0.0);
  result_cnc = new double(1.0);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute nrm2 on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDznrm2(handle, N, d_x, 1, result_cublas);

  // Compute nrm2 on GPU using cncblas
  *result_cnc = cncblasZnrm2(N, d_x);

  // Compare the results
  EXPECT_DOUBLE_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}
