#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(amin, singlePrecision) {
  float *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new float[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute amin on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIsamin(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amin on GPU using cncblas
  *result_cnc = cncblasSamin(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amin, doublePrecision) {
  double *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new double[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute amin on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIdamin(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amin on GPU using cncblas
  *result_cnc = cncblasDamin(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc) << INFINITY;

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amin, complexSinglePrecision) {
  hipComplex *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute amin on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIcamin(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amin on GPU using cncblas
  *result_cnc = cncblasCamin(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amin, complexDoublePrecision) {
  hipDoubleComplex *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute amin on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIzamin(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amin on GPU using cncblas
  *result_cnc = cncblasZamin(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}
