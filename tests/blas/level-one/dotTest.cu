#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(dot, singlePrecision) {
  float *h_x, *h_y;
  float *d_x, *d_y;
  float *result_cnc, *result_cublas;

  h_x = new float[N];
  h_y = new float[N];
  result_cublas = new float(0.0f);
  result_cnc = new float(1.0f);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
    h_y[i] = cncblasRandf;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSdot(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasSdot(N, d_x, d_y);

  // Compare the results
  EXPECT_NEAR(*result_cublas, *result_cnc, 1e-5);

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

TEST(dot, doublePrecision) {
  double *h_x, *h_y;
  double *d_x, *d_y;
  double *result_cnc, *result_cublas;

  h_x = new double[N];
  h_y = new double[N];
  result_cublas = new double(0.0);
  result_cnc = new double(1.0);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
    h_y[i] = cncblasRand;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDdot(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasDdot(N, d_x, d_y);

  // Compare the results
  EXPECT_NEAR(*result_cublas, *result_cnc, 1e-10);

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

TEST(dot, complexSinglePrecisionU) {
  hipComplex *h_x, *h_y;
  hipComplex *d_x, *d_y;
  hipComplex *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  h_y = new hipComplex[N];
  result_cublas = new hipComplex(make_hipComplex(0.0f, 0.0f));
  result_cnc = new hipComplex(make_hipComplex(1.0f, 0.0f));
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCdotu(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasCdotu(N, d_x, d_y);

  // Compare the results
  EXPECT_TRUE(cncblasComplexIsEqual(result_cublas, result_cnc))
                << "Expected: (" << result_cublas->x << "," << result_cublas->y << "); but got: (" << result_cnc->x
                << "," << result_cnc->y << ")" << std::endl << "Error: (" << std::abs(result_cublas->x - result_cnc->x)
                << "," << std::abs(result_cublas->y - result_cnc->y) << ")" << std::endl;

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

TEST(dot, complexSinglePrecisionC) {
  hipComplex *h_x, *h_y;
  hipComplex *d_x, *d_y;
  hipComplex *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  h_y = new hipComplex[N];
  result_cublas = new hipComplex(make_hipComplex(0.0f, 0.0f));
  result_cnc = new hipComplex(make_hipComplex(1.0f, 0.0f));
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCdotc(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasCdotc(N, d_x, d_y);

  // Compare the results
  EXPECT_TRUE(cncblasComplexIsEqual(result_cublas, result_cnc))
                << "Expected: (" << result_cublas->x << "," << result_cublas->y << "); but got: (" << result_cnc->x
                << "," << result_cnc->y << ")" << std::endl << "Error: (" << std::abs(result_cublas->x - result_cnc->x)
                << "," << std::abs(result_cublas->y - result_cnc->y) << ")" << std::endl;

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

TEST(dot, complexDoublePrecisionU) {
  hipDoubleComplex *h_x, *h_y;
  hipDoubleComplex *d_x, *d_y;
  hipDoubleComplex *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  h_y = new hipDoubleComplex[N];
  result_cublas = new hipDoubleComplex(make_hipDoubleComplex(0.0, 0.0));
  result_cnc = new hipDoubleComplex(make_hipDoubleComplex(1.0, 0.0));
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZdotu(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasZdotu(N, d_x, d_y);

  // Compare the results
  EXPECT_TRUE(cncblasComplexIsEqual(result_cublas, result_cnc))
                << "Expected: (" << result_cublas->x << "," << result_cublas->y << "); but got: (" << result_cnc->x
                << "," << result_cnc->y << ")" << std::endl << "Error: (" << std::abs(result_cublas->x - result_cnc->x)
                << "," << std::abs(result_cublas->y - result_cnc->y) << ")" << std::endl;

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

TEST(dot, complexDoublePrecisionC) {
  hipDoubleComplex *h_x, *h_y;
  hipDoubleComplex *d_x, *d_y;
  hipDoubleComplex *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  h_y = new hipDoubleComplex[N];
  result_cublas = new hipDoubleComplex(make_hipDoubleComplex(0.0, 0.0));
  result_cnc = new hipDoubleComplex(make_hipDoubleComplex(1.0, 0.0));
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute dot on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZdotc(handle, N, d_x, 1, d_y, 1, result_cublas);

  // Compute dot on GPU using cncblas
  *result_cnc = cncblasZdotc(N, d_x, d_y);

  // Compare the results
  EXPECT_TRUE(cncblasComplexIsEqual(result_cublas, result_cnc))
                << "Expected: (" << result_cublas->x << "," << result_cublas->y << "); but got: (" << result_cnc->x
                << "," << result_cnc->y << ")" << std::endl << "Error: (" << std::abs(result_cublas->x - result_cnc->x)
                << "," << std::abs(result_cublas->y - result_cnc->y) << ")" << std::endl;

  // Free memory
  delete[] h_x;
  delete[] h_y;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}
