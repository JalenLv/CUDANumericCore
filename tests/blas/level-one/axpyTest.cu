#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(axpy, singlePrecision) {
  float *alpha;
  float *h_x, *h_y_cnc, *h_y_cublas;
  float *d_x, *d_y_cnc, *d_y_cublas;

  alpha = new float(cncblasRandf);
  h_x = new float[N];
  h_y_cnc = new float[N];
  h_y_cublas = new float[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
    h_y_cnc[i] = cncblasRandf;
    h_y_cublas[i] = h_y_cnc[i];
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute axpy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSaxpy(handle, N, alpha, d_x, 1, d_y_cublas, 1);

  // Compute axpy on GPU using cncblas
  cncblasSaxpy(N, alpha, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(float), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_FLOAT_EQ(h_y_cublas[i], h_y_cnc[i]);
  }

  delete alpha;
  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(axpy, doublePrecision) {
  double *alpha;
  double *h_x, *h_y_cnc, *h_y_cublas;
  double *d_x, *d_y_cnc, *d_y_cublas;

  alpha = new double(cncblasRand);
  h_x = new double[N];
  h_y_cnc = new double[N];
  h_y_cublas = new double[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
    h_y_cnc[i] = cncblasRand;
    h_y_cublas[i] = h_y_cnc[i];
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute axpy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDaxpy(handle, N, alpha, d_x, 1, d_y_cublas, 1);

  // Compute axpy on GPU using cncblas
  cncblasDaxpy(N, alpha, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(double), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_DOUBLE_EQ(h_y_cublas[i], h_y_cnc[i]);
  }

  delete alpha;
  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(axpy, complexSinglePrecision) {
  hipComplex *alpha;
  hipComplex *h_x, *h_y_cnc, *h_y_cublas;
  hipComplex *d_x, *d_y_cnc, *d_y_cublas;

  alpha = new hipComplex;
  *alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  h_x = new hipComplex[N];
  h_y_cnc = new hipComplex[N];
  h_y_cublas = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y_cublas[i] = h_y_cnc[i];
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute axpy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCaxpy(handle, N, alpha, d_x, 1, d_y_cublas, 1);

  // Compute axpy on GPU using cncblas
  cncblasCaxpy(N, alpha, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cublas + i, h_y_cnc + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << "); but got: (" << h_y_cnc[i].x
                  << "," << h_y_cnc[i].y << ")" << std::endl << "Error: (" << std::abs(h_y_cublas[i].x - h_y_cnc[i].x)
                  << "," << std::abs(h_y_cublas[i].y - h_y_cnc[i].y) << ")" << std::endl;
  }

  delete alpha;
  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(axpy, complexDoublePrecision) {
  hipDoubleComplex *alpha;
  hipDoubleComplex *h_x, *h_y_cnc, *h_y_cublas;
  hipDoubleComplex *d_x, *d_y_cnc, *d_y_cublas;

  alpha = new hipDoubleComplex;
  *alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  h_x = new hipDoubleComplex[N];
  h_y_cnc = new hipDoubleComplex[N];
  h_y_cublas = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y_cublas[i] = h_y_cnc[i];
  }

  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute axpy on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZaxpy(handle, N, alpha, d_x, 1, d_y_cublas, 1);

  // Compute axpy on GPU using cncblas
  cncblasZaxpy(N, alpha, d_x, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cublas + i, h_y_cnc + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << "); but got: (" << h_y_cnc[i].x
                  << "," << h_y_cnc[i].y << ")" << std::endl << "Error: (" << std::abs(h_y_cublas[i].x - h_y_cnc[i].x)
                  << "," << std::abs(h_y_cublas[i].y - h_y_cnc[i].y) << ")" << std::endl;
  }

  delete alpha;
  delete[] h_x;
  delete[] h_y_cnc;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_y_cublas));
}

