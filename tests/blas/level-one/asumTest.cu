#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(asum, singlePrecision) {
  float *h_x, *d_x;
  float *result_cnc, *result_cublas;

  h_x = new float[N];
  result_cublas = new float(0.0f);
  result_cnc = new float(1.0f);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute asum on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSasum(handle, N, d_x, 1, result_cublas);

  // Compute asum on GPU using cncblas
  *result_cnc = cncblasSasum(N, d_x);

  // Compare the results
  EXPECT_FLOAT_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(asum, doublePrecision) {
  double *h_x, *d_x;
  double *result_cnc, *result_cublas;

  h_x = new double[N];
  result_cublas = new double(0.0);
  result_cnc = new double(1.0);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute asum on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDasum(handle, N, d_x, 1, result_cublas);

  // Compute asum on GPU using cncblas
  *result_cnc = cncblasDasum(N, d_x);

  // Compare the results
  EXPECT_DOUBLE_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(asum, complexSinglePrecision) {
  hipComplex *h_x, *d_x;
  float *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  result_cublas = new float(0.0f);
  result_cnc = new float(1.0f);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute asum on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasScasum(handle, N, d_x, 1, result_cublas);

  // Compute asum on GPU using cncblas
  *result_cnc = cncblasCasum(N, d_x);

  // Compare the results
  EXPECT_FLOAT_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(asum, complexDoublePrecision) {
  hipDoubleComplex *h_x, *d_x;
  double *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  result_cublas = new double(0.0);
  result_cnc = new double(1.0);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute asum on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDzasum(handle, N, d_x, 1, result_cublas);

  // Compute asum on GPU using cncblas
  *result_cnc = cncblasZasum(N, d_x);

  // Compare the results
  EXPECT_DOUBLE_EQ(*result_cublas, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}
