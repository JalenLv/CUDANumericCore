#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int N = 1 << 12;

TEST(scal, singlePrecision) {
  float *h_alpha, *h_x_cnc, *h_x_cublas;
  float *d_x_cnc, *d_x_cublas;

  h_x_cnc = new float[N];
  h_x_cublas = new float[N];
  h_alpha = new float;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRandf;
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = cncblasRandf;
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(float), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasSscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(float), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_FLOAT_EQ(h_x_cnc[i], h_x_cublas[i]);
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}

TEST(scal, doublePrecision) {
  double *h_alpha, *h_x_cnc, *h_x_cublas;
  double *d_x_cnc, *d_x_cublas;

  h_x_cnc = new double[N];
  h_x_cublas = new double[N];
  h_alpha = new double;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRand;
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = cncblasRand;
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(double), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasDscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(double), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_DOUBLE_EQ(h_x_cnc[i], h_x_cublas[i]);
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}

TEST(scal, complexSinglePrecision) {
  hipComplex *h_alpha, *h_x_cnc, *h_x_cublas;
  hipComplex *d_x_cnc, *d_x_cublas;

  h_x_cnc = new hipComplex[N];
  h_x_cublas = new hipComplex[N];
  h_alpha = new hipComplex;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasCscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x_cnc + i, h_x_cublas + i));
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete[] h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}

TEST(scal, complexDoublePrecision) {
  hipDoubleComplex *h_alpha, *h_x_cnc, *h_x_cublas;
  hipDoubleComplex *d_x_cnc, *d_x_cublas;

  h_x_cnc = new hipDoubleComplex[N];
  h_x_cublas = new hipDoubleComplex[N];
  h_alpha = new hipDoubleComplex;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasZscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x_cnc + i, h_x_cublas + i));
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete[] h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}

TEST(scal, complexSinglePrecisionReal) {
  float *h_alpha;
  hipComplex *h_x_cnc, *h_x_cublas;
  hipComplex *d_x_cnc, *d_x_cublas;

  h_x_cnc = new hipComplex[N];
  h_x_cublas = new hipComplex[N];
  h_alpha = new float;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = cncblasRandf;
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCsscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasCsscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x_cnc + i, h_x_cublas + i));
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}

TEST(scal, complexDoublePrecisionReal) {
  double *h_alpha;
  hipDoubleComplex *h_x_cnc, *h_x_cublas;
  hipDoubleComplex *d_x_cnc, *d_x_cublas;

  h_x_cnc = new hipDoubleComplex[N];
  h_x_cublas = new hipDoubleComplex[N];
  h_alpha = new double;
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_x_cublas[i] = h_x_cnc[i];
  }
  *h_alpha = cncblasRand;
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cublas, h_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform scal using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZdscal(handle, N, h_alpha, d_x_cublas, 1);

  // Perform scal using cncblas
  cncblasZdscal(N, h_alpha, d_x_cnc);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_x_cnc, d_x_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_x_cublas, d_x_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_x_cnc + i, h_x_cublas + i));
  }

  delete[] h_x_cnc;
  delete[] h_x_cublas;
  delete h_alpha;
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_x_cublas));
}
