#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cncblas.h>

const int N = 1 << 10;

TEST(amax, singlePrecision) {
  float *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new float[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = rand() / (float) RAND_MAX;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute amax on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIsamax(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amax on GPU using cncblas
  *result_cnc = cncblasSamax(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amax, doublePrecision) {
  double *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new double[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = rand() / (double) RAND_MAX;
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute amax on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIdamax(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amax on GPU using cncblas
  *result_cnc = cncblasDamax(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amax, complexSinglePrecision) {
  hipComplex *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new hipComplex[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(rand() / (float) RAND_MAX, rand() / (float) RAND_MAX);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute amax on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIcamax(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amax on GPU using cncblas
  *result_cnc = cncblasCamax(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}

TEST(amax, complexDoublePrecision) {
  hipDoubleComplex *h_x, *d_x;
  size_t *result_cnc, *result_cublas;

  h_x = new hipDoubleComplex[N];
  result_cublas = new size_t(0);
  result_cnc = new size_t(1);
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(rand() / (double) RAND_MAX, rand() / (double) RAND_MAX);
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute amax on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasIzamax(handle, N, d_x, 1, reinterpret_cast<int *>(result_cublas));

  // Compute amax on GPU using cncblas
  *result_cnc = cncblasZamax(N, d_x);

  // Compare the results
  EXPECT_EQ(*result_cublas - 1, *result_cnc);

  // Free memory
  delete[] h_x;
  delete result_cublas;
  delete result_cnc;
  checkCudaErrors(hipFree(d_x));
}
