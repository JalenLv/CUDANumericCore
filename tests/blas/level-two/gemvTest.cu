#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int M = 1 << 10;
const int N = 1 << 8;
const double epsilon = 1e-5;

TEST(gemv, singlePrecisionN) {
  float *alpha, *beta;
  float *h_x, *d_x;
  // row major - cncblas
  float *h_A_cnc, *h_y_cnc;
  float *d_A_cnc, *d_y_cnc;
  // col major - cublas
  float *h_A_cublas, *h_y_cublas;
  float *d_A_cublas, *d_y_cublas;

  alpha = new float;
  beta = new float;
  h_x = new float[N];
  h_A_cnc = new float[M * N];
  h_y_cnc = new float[M];
  h_A_cublas = new float[M * N];
  h_y_cublas = new float[M];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cublas, M * sizeof(float)));

  srand(time(NULL));
  *alpha = cncblasRandf;
  *beta = cncblasRandf;
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = cncblasRandf;
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRandf;
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRandf;
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, M * sizeof(float), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemv(handle, HIPBLAS_OP_N, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasSgemv(CNCBLAS_OP_N, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, M * sizeof(float), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < M; i++) {
    EXPECT_NEAR(h_y_cublas[i], h_y_cnc[i], epsilon);
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, doublePrecisionN) {
  double *alpha, *beta;
  double *h_x, *d_x;
  // row major - cncblas
  double *h_A_cnc, *h_y_cnc;
  double *d_A_cnc, *d_y_cnc;
  // col major - cublas
  double *h_A_cublas, *h_y_cublas;
  double *d_A_cublas, *d_y_cublas;

  alpha = new double;
  beta = new double;
  h_x = new double[N];
  h_A_cnc = new double[M * N];
  h_y_cnc = new double[M];
  h_A_cublas = new double[M * N];
  h_y_cublas = new double[M];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cublas, M * sizeof(double)));

  srand(time(NULL));
  *alpha = cncblasRand;
  *beta = cncblasRand;
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = cncblasRand;
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x[i] = cncblasRand;
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRand;
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, M * sizeof(double), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDgemv(handle, HIPBLAS_OP_N, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasDgemv(CNCBLAS_OP_N, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, M * sizeof(double), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < M; i++) {
    EXPECT_NEAR(h_y_cublas[i], h_y_cnc[i], epsilon);
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexSinglePrecisionN) {
  hipComplex *alpha, *beta;
  hipComplex *h_x, *d_x;
  // row major - cncblas
  hipComplex *h_A_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipComplex *h_A_cublas, *h_y_cublas;
  hipComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipComplex;
  beta = new hipComplex;
  h_x = new hipComplex[N];
  h_A_cnc = new hipComplex[M * N];
  h_y_cnc = new hipComplex[M];
  h_A_cublas = new hipComplex[M * N];
  h_y_cublas = new hipComplex[M];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, M * sizeof(hipComplex)));

  srand(time(NULL));
  *alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  *beta = make_hipComplex(cncblasRandf, cncblasRandf);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipComplex(cncblasRandf, cncblasRandf);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, M * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCgemv(handle, HIPBLAS_OP_N, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasCgemv(CNCBLAS_OP_N, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, M * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < M; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexDoublePrecisionN) {
  hipDoubleComplex *alpha, *beta;
  hipDoubleComplex *h_x, *d_x;
  // row major - cncblas
  hipDoubleComplex *h_A_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipDoubleComplex *h_A_cublas, *h_y_cublas;
  hipDoubleComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipDoubleComplex;
  beta = new hipDoubleComplex;
  h_x = new hipDoubleComplex[N];
  h_A_cnc = new hipDoubleComplex[M * N];
  h_y_cnc = new hipDoubleComplex[M];
  h_A_cublas = new hipDoubleComplex[M * N];
  h_y_cublas = new hipDoubleComplex[M];
  checkCudaErrors(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, M * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  *alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  *beta = make_hipDoubleComplex(cncblasRand, cncblasRand);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipDoubleComplex(cncblasRand, cncblasRand);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZgemv(handle, HIPBLAS_OP_N, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasZgemv(CNCBLAS_OP_N, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < M; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, singlePrecisionT) {
  float *alpha, *beta;
  float *h_x, *d_x;
  // row major - cncblas
  float *h_A_cnc, *h_y_cnc;
  float *d_A_cnc, *d_y_cnc;
  // col major - cublas
  float *h_A_cublas, *h_y_cublas;
  float *d_A_cublas, *d_y_cublas;

  alpha = new float;
  beta = new float;
  h_x = new float[M];
  h_A_cnc = new float[M * N];
  h_y_cnc = new float[N];
  h_A_cublas = new float[M * N];
  h_y_cublas = new float[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(float)));

  srand(time(NULL));
  *alpha = cncblasRandf;
  *beta = cncblasRandf;
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = cncblasRandf;
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = cncblasRandf;
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandf;
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(float), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemv(handle, HIPBLAS_OP_T, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasSgemv(CNCBLAS_OP_T, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(float), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_NEAR(h_y_cublas[i], h_y_cnc[i], epsilon);
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, doublePrecisionT) {
  double *alpha, *beta;
  double *h_x, *d_x;
  // row major - cncblas
  double *h_A_cnc, *h_y_cnc;
  double *d_A_cnc, *d_y_cnc;
  // col major - cublas
  double *h_A_cublas, *h_y_cublas;
  double *d_A_cublas, *d_y_cublas;

  alpha = new double;
  beta = new double;
  h_x = new double[M];
  h_A_cnc = new double[M * N];
  h_y_cnc = new double[N];
  h_A_cublas = new double[M * N];
  h_y_cublas = new double[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(double)));

  srand(time(NULL));
  *alpha = cncblasRand;
  *beta = cncblasRand;
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = cncblasRand;
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = cncblasRand;
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRand;
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(double), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDgemv(handle, HIPBLAS_OP_T, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasDgemv(CNCBLAS_OP_T, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(double), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_NEAR(h_y_cublas[i], h_y_cnc[i], epsilon);
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexSinglePrecisionT) {
  hipComplex *alpha, *beta;
  hipComplex *h_x, *d_x;
  // row major - cncblas
  hipComplex *h_A_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipComplex *h_A_cublas, *h_y_cublas;
  hipComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipComplex;
  beta = new hipComplex;
  h_x = new hipComplex[M];
  h_A_cnc = new hipComplex[M * N];
  h_y_cnc = new hipComplex[N];
  h_A_cublas = new hipComplex[M * N];
  h_y_cublas = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  *alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  *beta = make_hipComplex(cncblasRandf, cncblasRandf);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipComplex(cncblasRandf, cncblasRandf);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCgemv(handle, HIPBLAS_OP_T, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasCgemv(CNCBLAS_OP_T, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexDoublePrecisionT) {
  hipDoubleComplex *alpha, *beta;
  hipDoubleComplex *h_x, *d_x;
  // row major - cncblas
  hipDoubleComplex *h_A_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipDoubleComplex *h_A_cublas, *h_y_cublas;
  hipDoubleComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipDoubleComplex;
  beta = new hipDoubleComplex;
  h_x = new hipDoubleComplex[M];
  h_A_cnc = new hipDoubleComplex[M * N];
  h_y_cnc = new hipDoubleComplex[N];
  h_A_cublas = new hipDoubleComplex[M * N];
  h_y_cublas = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  *alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  *beta = make_hipDoubleComplex(cncblasRand, cncblasRand);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipDoubleComplex(cncblasRand, cncblasRand);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZgemv(handle, HIPBLAS_OP_T, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasZgemv(CNCBLAS_OP_T, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexSinglePrecisionC) {
  hipComplex *alpha, *beta;
  hipComplex *h_x, *d_x;
  // row major - cncblas
  hipComplex *h_A_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipComplex *h_A_cublas, *h_y_cublas;
  hipComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipComplex;
  beta = new hipComplex;
  h_x = new hipComplex[M];
  h_A_cnc = new hipComplex[M * N];
  h_y_cnc = new hipComplex[N];
  h_A_cublas = new hipComplex[M * N];
  h_y_cublas = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipComplex)));

  srand(time(NULL));
  *alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  *beta = make_hipComplex(cncblasRandf, cncblasRandf);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipComplex(cncblasRandf, cncblasRandf);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = make_hipComplex(cncblasRandf, cncblasRandf);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCgemv(handle, HIPBLAS_OP_C, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasCgemv(CNCBLAS_OP_C, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}

TEST(gemv, complexDoublePrecisionC) {
  hipDoubleComplex *alpha, *beta;
  hipDoubleComplex *h_x, *d_x;
  // row major - cncblas
  hipDoubleComplex *h_A_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_y_cnc;
  // col major - cublas
  hipDoubleComplex *h_A_cublas, *h_y_cublas;
  hipDoubleComplex *d_A_cublas, *d_y_cublas;

  alpha = new hipDoubleComplex;
  beta = new hipDoubleComplex;
  h_x = new hipDoubleComplex[M];
  h_A_cnc = new hipDoubleComplex[M * N];
  h_y_cnc = new hipDoubleComplex[N];
  h_A_cublas = new hipDoubleComplex[M * N];
  h_y_cublas = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cnc, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));

  srand(time(NULL));
  *alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  *beta = make_hipDoubleComplex(cncblasRand, cncblasRand);
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cnc[row * N + col] = make_hipDoubleComplex(cncblasRand, cncblasRand);
      h_A_cublas[col * M + row] = h_A_cnc[row * N + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
    h_y_cublas[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cublas, h_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Compute gemv on GPU using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZgemv(handle, HIPBLAS_OP_C, M, N, alpha, d_A_cublas, M, d_x, 1, beta, d_y_cublas, 1);

  // Compute gemv on GPU using cncblas
  cncblasZgemv(CNCBLAS_OP_C, M, N, alpha, d_A_cnc, d_x, beta, d_y_cnc);

  // Copy the results back to host
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_cublas, d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Compare the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_cublas + i))
                  << "Expected: (" << h_y_cublas[i].x << "," << h_y_cublas[i].y << ")\nGot: ("
                  << h_y_cnc[i].x << "," << h_y_cnc[i].y << ")\nError: (" << std::abs(h_y_cnc[i].x - h_y_cublas[i].x)
                  << "," << std::abs(h_y_cnc[i].y - h_y_cublas[i].y) << ")";
  }

  delete alpha;
  delete beta;
  delete[] h_x;
  delete[] h_A_cnc;
  delete[] h_y_cnc;
  delete[] h_A_cublas;
  delete[] h_y_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_cublas));
  checkCudaErrors(hipFree(d_y_cublas));
}
