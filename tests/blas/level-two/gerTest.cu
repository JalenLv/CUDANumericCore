#include <gtest/gtest.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int M = 1 << 10;
const int N = 1 << 8;
const double epsilon = 1e-5;

TEST(ger, singlePrecision) {
  float *h_alpha, *h_x, *h_y;
  float *d_x, *d_y;
  // row major - cncblas
  float *h_A_cncblas;
  float *d_A_cncblas;
  // column major - cublas
  float *h_A_cublas;
  float *d_A_cublas;

  h_alpha = new float;
  h_x = new float[M];
  h_y = new float[N];
  h_A_cncblas = new float[M * N];
  h_A_cublas = new float[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(float)));

  *h_alpha = cncblasRandf;
  for (int i = 0; i < M; i++) {
    h_x[i] = cncblasRandf;
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = cncblasRandf;
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = cncblasRandf;
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(float), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSger(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasSger(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(float), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_NEAR(h_A_cublas[col * M + row], h_A_cncblas[row * N + col], epsilon);
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}

TEST(ger, doublePrecision) {
  double *h_alpha, *h_x, *h_y;
  double *d_x, *d_y;
  // row major - cncblas
  double *h_A_cncblas;
  double *d_A_cncblas;
  // column major - cublas
  double *h_A_cublas;
  double *d_A_cublas;

  h_alpha = new double;
  h_x = new double[M];
  h_y = new double[N];
  h_A_cncblas = new double[M * N];
  h_A_cublas = new double[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(double)));

  *h_alpha = cncblasRand;
  for (int i = 0; i < M; i++) {
    h_x[i] = cncblasRand;
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = cncblasRand;
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = cncblasRand;
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(double), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDger(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasDger(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(double), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_NEAR(h_A_cublas[col * M + row], h_A_cncblas[row * N + col], epsilon);
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}

TEST(ger, singlePrecisionComplexU) {
  hipComplex *h_alpha, *h_x, *h_y;
  hipComplex *d_x, *d_y;
  // row major - cncblas
  hipComplex *h_A_cncblas;
  hipComplex *d_A_cncblas;
  // column major - cublas
  hipComplex *h_A_cublas;
  hipComplex *d_A_cublas;

  h_alpha = new hipComplex;
  h_x = new hipComplex[M];
  h_y = new hipComplex[N];
  h_A_cncblas = new hipComplex[M * N];
  h_A_cublas = new hipComplex[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipComplex)));

  *h_alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = make_hipComplex(cncblasRandf, cncblasRandf);
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCgeru(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasCgeru(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_TRUE(cncblasComplexIsEqual(h_A_cncblas + row * N + col, h_A_cublas + col * M + row));
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}

TEST(ger, doublePrecisionComplexU) {
  hipDoubleComplex *h_alpha, *h_x, *h_y;
  hipDoubleComplex *d_x, *d_y;
  // row major - cncblas
  hipDoubleComplex *h_A_cncblas;
  hipDoubleComplex *d_A_cncblas;
  // column major - cublas
  hipDoubleComplex *h_A_cublas;
  hipDoubleComplex *d_A_cublas;

  h_alpha = new hipDoubleComplex;
  h_x = new hipDoubleComplex[M];
  h_y = new hipDoubleComplex[N];
  h_A_cncblas = new hipDoubleComplex[M * N];
  h_A_cublas = new hipDoubleComplex[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipDoubleComplex)));

  *h_alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = make_hipDoubleComplex(cncblasRand, cncblasRand);
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZgeru(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasZgeru(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_TRUE(cncblasComplexIsEqual(h_A_cncblas + row * N + col, h_A_cublas + col * M + row));
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}

TEST(ger, singlePrecisionComplexC) {
  hipComplex *h_alpha, *h_x, *h_y;
  hipComplex *d_x, *d_y;
  // row major - cncblas
  hipComplex *h_A_cncblas;
  hipComplex *d_A_cncblas;
  // column major - cublas
  hipComplex *h_A_cublas;
  hipComplex *d_A_cublas;

  h_alpha = new hipComplex;
  h_x = new hipComplex[M];
  h_y = new hipComplex[N];
  h_A_cncblas = new hipComplex[M * N];
  h_A_cublas = new hipComplex[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipComplex)));

  *h_alpha = make_hipComplex(cncblasRandf, cncblasRandf);
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = make_hipComplex(cncblasRandf, cncblasRandf);
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = make_hipComplex(cncblasRandf, cncblasRandf);
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasCgerc(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasCgerc(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_TRUE(cncblasComplexIsEqual(h_A_cncblas + row * N + col, h_A_cublas + col * M + row));
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}

TEST(ger, doublePrecisionComplexC) {
  hipDoubleComplex *h_alpha, *h_x, *h_y;
  hipDoubleComplex *d_x, *d_y;
  // row major - cncblas
  hipDoubleComplex *h_A_cncblas;
  hipDoubleComplex *d_A_cncblas;
  // column major - cublas
  hipDoubleComplex *h_A_cublas;
  hipDoubleComplex *d_A_cublas;

  h_alpha = new hipDoubleComplex;
  h_x = new hipDoubleComplex[M];
  h_y = new hipDoubleComplex[N];
  h_A_cncblas = new hipDoubleComplex[M * N];
  h_A_cublas = new hipDoubleComplex[M * N];
  checkCudaErrors(hipMalloc(&d_x, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cncblas, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_A_cublas, M * N * sizeof(hipDoubleComplex)));

  *h_alpha = make_hipDoubleComplex(cncblasRand, cncblasRand);
  for (int i = 0; i < M; i++) {
    h_x[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int i = 0; i < N; i++) {
    h_y[i] = make_hipDoubleComplex(cncblasRand, cncblasRand);
  }
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      h_A_cncblas[row * N + col] = make_hipDoubleComplex(cncblasRand, cncblasRand);
      h_A_cublas[col * M + row] = h_A_cncblas[row * N + col];
    }
  }
  checkCudaErrors(hipMemcpy(d_x, h_x, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y, h_y, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cncblas, h_A_cncblas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_cublas, h_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Computer ger using cublas
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasZgerc(handle, M, N, h_alpha, d_x, 1, d_y, 1, d_A_cublas, M);

  // Compute ger using cncblas
  cncblasZgerc(M, N, h_alpha, d_x, d_y, d_A_cncblas);

  // Copy the result from device to host
  checkCudaErrors(hipMemcpy(h_A_cublas, d_A_cublas, M * N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_A_cncblas, d_A_cncblas, M * N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the result
  for (int row = 0; row < M; row++) {
    for (int col = 0; col < N; col++) {
      EXPECT_TRUE(cncblasComplexIsEqual(h_A_cncblas + row * N + col, h_A_cublas + col * M + row));
    }
  }

  // Free the memory
  delete h_alpha;
  delete h_x;
  delete h_y;
  delete h_A_cncblas;
  delete h_A_cublas;
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
  checkCudaErrors(hipFree(d_A_cncblas));
  checkCudaErrors(hipFree(d_A_cublas));
}
