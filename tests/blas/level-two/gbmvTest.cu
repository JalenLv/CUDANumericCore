#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "cncblas.h"

const int M = 1 << 10;
const int N = 1 << 11;
const int kl = 100;
const int ku = 120;
const double epsilon = 1e-5;

TEST(gbmv, singlePrecisionN) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  float *alpha = new float(cncblasRandf);
  float *beta = new float(cncblasRandf);

  // cncblas - 0 based
  float *h_A_cnc, *h_x_cnc, *h_y_cnc;
  float *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new float[nColsA * nRowsA];
  h_x_cnc = new float[N];
  h_y_cnc = new float[M];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(float)));

  // using gemv to verify the correctness of the cncblas implementation
  float *h_A_gemv, *h_x_gemv, *h_y_gemv;
  float *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new float[M * N];
  h_x_gemv = new float[N];
  h_y_gemv = new float[M];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_gemv, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_gemv, M * sizeof(float)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(float));
  memset(h_A_gemv, 0, M * N * sizeof(float));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandf;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRandf;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRandf;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, M * sizeof(float), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasSgbmv(CNCBLAS_OP_N, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasSgemv(CNCBLAS_OP_N, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, M * sizeof(float), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < M; i++) {
    EXPECT_NEAR(h_y_cnc[i], h_y_gemv[i], epsilon) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, singlePrecisionT) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  float *alpha = new float(cncblasRandf);
  float *beta = new float(cncblasRandf);

  // cncblas - 0 based
  float *h_A_cnc, *h_x_cnc, *h_y_cnc;
  float *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new float[nColsA * nRowsA];
  h_x_cnc = new float[M];
  h_y_cnc = new float[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(float)));

  // using gemv to verify the correctness of the cncblas implementation
  float *h_A_gemv, *h_x_gemv, *h_y_gemv;
  float *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new float[M * N];
  h_x_gemv = new float[M];
  h_y_gemv = new float[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(float)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(float));
  memset(h_A_gemv, 0, M * N * sizeof(float));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandf;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRandf;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandf;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(float), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(float), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasSgbmv(CNCBLAS_OP_T, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasSgemv(CNCBLAS_OP_T, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(float), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_NEAR(h_y_cnc[i], h_y_gemv[i], epsilon) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, doublePrecisionN) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  double *alpha = new double(cncblasRand);
  double *beta = new double(cncblasRand);

  // cncblas - 0 based
  double *h_A_cnc, *h_x_cnc, *h_y_cnc;
  double *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new double[nColsA * nRowsA];
  h_x_cnc = new double[N];
  h_y_cnc = new double[M];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(double)));

  // using gemv to verify the correctness of the cncblas implementation
  double *h_A_gemv, *h_x_gemv, *h_y_gemv;
  double *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new double[M * N];
  h_x_gemv = new double[N];
  h_y_gemv = new double[M];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_gemv, N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_gemv, M * sizeof(double)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(double));
  memset(h_A_gemv, 0, M * N * sizeof(double));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRand;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRand;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRand;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, M * sizeof(double), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasDgbmv(CNCBLAS_OP_N, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasDgemv(CNCBLAS_OP_N, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, M * sizeof(double), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < M; i++) {
    EXPECT_NEAR(h_y_cnc[i], h_y_gemv[i], epsilon) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, doublePrecisionT) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  double *alpha = new double(cncblasRand);
  double *beta = new double(cncblasRand);

  // cncblas - 0 based
  double *h_A_cnc, *h_x_cnc, *h_y_cnc;
  double *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new double[nColsA * nRowsA];
  h_x_cnc = new double[M];
  h_y_cnc = new double[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(double)));

  // using gemv to verify the correctness of the cncblas implementation
  double *h_A_gemv, *h_x_gemv, *h_y_gemv;
  double *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new double[M * N];
  h_x_gemv = new double[M];
  h_y_gemv = new double[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(double)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(double));
  memset(h_A_gemv, 0, M * N * sizeof(double));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRand;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRand;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRand;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(double), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(double), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasDgbmv(CNCBLAS_OP_T, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasDgemv(CNCBLAS_OP_T, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(double), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_NEAR(h_y_cnc[i], h_y_gemv[i], epsilon) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexSinglePrecisionN) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipComplex *alpha = new hipComplex(cncblasRandC);
  hipComplex *beta = new hipComplex(cncblasRandC);

  // cncblas - 0 based
  hipComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipComplex[nColsA * nRowsA];
  h_x_cnc = new hipComplex[N];
  h_y_cnc = new hipComplex[M];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(hipComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipComplex[M * N];
  h_x_gemv = new hipComplex[N];
  h_y_gemv = new hipComplex[M];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, M * sizeof(hipComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandC;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRandC;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRandC;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, M * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasCgbmv(CNCBLAS_OP_N, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasCgemv(CNCBLAS_OP_N, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, M * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < M; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i << "\n"
                                                                        << "Expected: " << h_y_gemv[i].x << " + " << h_y_gemv[i].y << "i\n"
                                                                        << " Got: " << h_y_cnc[i].x << " + " << h_y_cnc[i].y << "i\n"
                                                                        << "Error: " << h_y_cnc[i].x - h_y_gemv[i].x << " + " << h_y_cnc[i].y - h_y_gemv[i].y << "i";
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexSinglePrecisionT) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipComplex *alpha = new hipComplex(cncblasRandC);
  hipComplex *beta = new hipComplex(cncblasRandC);

  // cncblas - 0 based
  hipComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipComplex[nColsA * nRowsA];
  h_x_cnc = new hipComplex[M];
  h_y_cnc = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipComplex[M * N];
  h_x_gemv = new hipComplex[M];
  h_y_gemv = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(hipComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandC;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRandC;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandC;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasCgbmv(CNCBLAS_OP_T, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasCgemv(CNCBLAS_OP_T, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexSinglePrecisionC) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipComplex *alpha = new hipComplex(cncblasRandC);
  hipComplex *beta = new hipComplex(cncblasRandC);

  // cncblas - 0 based
  hipComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipComplex[nColsA * nRowsA];
  h_x_cnc = new hipComplex[M];
  h_y_cnc = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipComplex[M * N];
  h_x_gemv = new hipComplex[M];
  h_y_gemv = new hipComplex[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(hipComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(hipComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandC;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRandC;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandC;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(hipComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(hipComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasCgbmv(CNCBLAS_OP_C, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasCgemv(CNCBLAS_OP_C, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(hipComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexDoublePrecisionN) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipDoubleComplex *alpha = new hipDoubleComplex(cncblasRandZ);
  hipDoubleComplex *beta = new hipDoubleComplex(cncblasRandZ);

  // cncblas - 0 based
  hipDoubleComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipDoubleComplex[nColsA * nRowsA];
  h_x_cnc = new hipDoubleComplex[N];
  h_y_cnc = new hipDoubleComplex[M];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, M * sizeof(hipDoubleComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipDoubleComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipDoubleComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipDoubleComplex[M * N];
  h_x_gemv = new hipDoubleComplex[N];
  h_y_gemv = new hipDoubleComplex[M];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, M * sizeof(hipDoubleComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipDoubleComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipDoubleComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandZ;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < N; i++) {
    h_x_cnc[i] = cncblasRandZ;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < M; i++) {
    h_y_cnc[i] = cncblasRandZ;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasZgbmv(CNCBLAS_OP_N, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasZgemv(CNCBLAS_OP_N, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < M; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i << "\n"
                                                                  << "Expected: " << h_y_gemv[i].x << " + " << h_y_gemv[i].y << "i\n"
                                                                  << " Got: " << h_y_cnc[i].x << " + " << h_y_cnc[i].y << "i\n"
                                                                  << "Error: " << h_y_cnc[i].x - h_y_gemv[i].x << " + " << h_y_cnc[i].y - h_y_gemv[i].y << "i";
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexDoublePrecisionT) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipDoubleComplex *alpha = new hipDoubleComplex(cncblasRandZ);
  hipDoubleComplex *beta = new hipDoubleComplex(cncblasRandZ);

  // cncblas - 0 based
  hipDoubleComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipDoubleComplex[nColsA * nRowsA];
  h_x_cnc = new hipDoubleComplex[M];
  h_y_cnc = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipDoubleComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipDoubleComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipDoubleComplex[M * N];
  h_x_gemv = new hipDoubleComplex[M];
  h_y_gemv = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(hipDoubleComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipDoubleComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipDoubleComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandZ;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRandZ;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandZ;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasZgbmv(CNCBLAS_OP_T, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasZgemv(CNCBLAS_OP_T, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}

TEST(gbmv, complexDoublePrecisionC) {
  int nColsA = cncblasMin(N, M + ku);
  int nRowsA = ku + kl + 1;
  hipDoubleComplex *alpha = new hipDoubleComplex(cncblasRandZ);
  hipDoubleComplex *beta = new hipDoubleComplex(cncblasRandZ);

  // cncblas - 0 based
  hipDoubleComplex *h_A_cnc, *h_x_cnc, *h_y_cnc;
  hipDoubleComplex *d_A_cnc, *d_x_cnc, *d_y_cnc;
  h_A_cnc = new hipDoubleComplex[nColsA * nRowsA];
  h_x_cnc = new hipDoubleComplex[M];
  h_y_cnc = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_cnc, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_cnc, N * sizeof(hipDoubleComplex)));

  // using gemv to verify the correctness of the cncblas implementation
  hipDoubleComplex *h_A_gemv, *h_x_gemv, *h_y_gemv;
  hipDoubleComplex *d_A_gemv, *d_x_gemv, *d_y_gemv;
  h_A_gemv = new hipDoubleComplex[M * N];
  h_x_gemv = new hipDoubleComplex[M];
  h_y_gemv = new hipDoubleComplex[N];
  checkCudaErrors(hipMalloc(&d_A_gemv, M * N * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_x_gemv, M * sizeof(hipDoubleComplex)));
  checkCudaErrors(hipMalloc(&d_y_gemv, N * sizeof(hipDoubleComplex)));

  memset(h_A_cnc, 0, nColsA * nRowsA * sizeof(hipDoubleComplex));
  memset(h_A_gemv, 0, M * N * sizeof(hipDoubleComplex));
  for (int col = 0; col < nColsA; col++) {
    for (int row = cncblasMax(0, col - ku); row <= cncblasMin(M - 1, col + kl); row++) {
      h_A_cnc[(row - col + ku) * nColsA + col] = cncblasRandZ;
      h_A_gemv[row * N + col] = h_A_cnc[(row - col + ku) * nColsA + col];
    }
  }
  for (int i = 0; i < M; i++) {
    h_x_cnc[i] = cncblasRandZ;
    h_x_gemv[i] = h_x_cnc[i];
  }
  for (int i = 0; i < N; i++) {
    h_y_cnc[i] = cncblasRandZ;
    h_y_gemv[i] = h_y_cnc[i];
  }
  checkCudaErrors(hipMemcpy(d_A_cnc, h_A_cnc, nColsA * nRowsA * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_cnc, h_x_cnc, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_cnc, h_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_A_gemv, h_A_gemv, M * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_x_gemv, h_x_gemv, M * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_y_gemv, h_y_gemv, N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  // Perform gbmv using cncblas
  cncblasZgbmv(CNCBLAS_OP_C, M, N, kl, ku, alpha, d_A_cnc, d_x_cnc, beta, d_y_cnc);

  // Verify the results using gemv
  cncblasZgemv(CNCBLAS_OP_C, M, N, alpha, d_A_gemv, d_x_gemv, beta, d_y_gemv);

  // Copy the results back
  checkCudaErrors(hipMemcpy(h_y_cnc, d_y_cnc, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_y_gemv, d_y_gemv, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));

  // Check the results
  for (int i = 0; i < N; i++) {
    EXPECT_TRUE(cncblasComplexIsEqual(h_y_cnc + i, h_y_gemv + i)) << "at index " << i;
  }

  // Free the memory
  delete[] h_A_cnc;
  delete[] h_x_cnc;
  delete[] h_y_cnc;
  delete[] h_A_gemv;
  delete[] h_x_gemv;
  delete[] h_y_gemv;
  delete alpha;
  delete beta;
  checkCudaErrors(hipFree(d_A_cnc));
  checkCudaErrors(hipFree(d_x_cnc));
  checkCudaErrors(hipFree(d_y_cnc));
  checkCudaErrors(hipFree(d_A_gemv));
  checkCudaErrors(hipFree(d_x_gemv));
  checkCudaErrors(hipFree(d_y_gemv));
}
